#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <typeinfo>
#include <limits>
#include "optflow_gpu3d_reshape.h"
#include "../Derivatives/smoothnessterm_gpu3d_rs.cuh"
#include "gpu_constants.cuh"

/*********************************************************************************************************************************************************
 * Location: Helmholtz-Zentrum fuer Material und Kuestenforschung, Max-Planck-Strasse 1, 21502 Geesthacht
 * Author: Stefan Bruns
 * Contact: bruns@nano.ku.dk
 *
 * License: TBA
 *********************************************************************************************************************************************************/

namespace optflow
{
	namespace gpu3d_rs
	{
		__device__ __inline__ float interpolate_cubic(float &y0, float &y1, float &y2, float &y3, float &mu)
		{
			float mu2 = mu*mu;

			float a0 = y3-y2-y0+y1;
			float a1 = y0-y1-a0;
			float a2 = y2-y0;
			float a3 = y1;

			return a0*mu*mu2+a1*mu2+a2*mu+a3;
		}

		__global__ void gaussianfilter3D_x(optflow_type *input, optflow_type *output)
		{
			//acquire constants
			/////////////////////////////////////////////
			float sigma = gpu_const::filter_sigma_c;

			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			idx_type n_even = nstack-(nstack/2);

			bool outofbounds = false;
			idx_type idx = (blockIdx.x*blockDim.x+threadIdx.x);
			if (idx >= nstack) {outofbounds = true; idx = threadIdx.x;}

			idx_type pos = idx2pos3D(idx, nx, nslice, n_even);
			int z = pos/nslice;
			int y = (pos-z*nslice)/nx;
			int x = pos-z*nslice-y*nx;
			/////////////////////////////////////////////

			//Create Gaussian kernel
			///////////////////////////////////////////////////
			int fsize = (int) (3*sigma);
			float kernelsum = 0.0f;
			float valuesum = 0.0f;
			//////////////////////////////////////////////////

			for(int xi=-fsize; xi<=fsize; xi++)
			{
				int x0 = x+xi;

				//reflective boundaries
				if (x0 < 0) x0 = -x0;
				else if (x0 >= nx) x0 = 2*nx-x0-2;

				float kernel_val = expf(-(xi*xi)/(sigma*sigma*2));
				kernelsum += kernel_val;

				idx_type idx0 = xyz2idx(x0, y, z, nx, nslice, n_even);

				__syncthreads();
				if (x0 < 0 || x0 >= nx) continue;
				valuesum += kernel_val*input[idx0];
			}

			if(!outofbounds)
				output[idx] = valuesum/kernelsum;

			return;
		}
		__global__ void gaussianfilter3D_y(optflow_type *input, optflow_type *output)
		{
			//acquire constants
			/////////////////////////////////////////////
			float sigma = gpu_const::filter_sigma_c;

			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			idx_type n_even = nstack-(nstack/2);

			bool outofbounds = false;
			idx_type idx = (blockIdx.x*blockDim.x+threadIdx.x);
			if (idx >= nstack) {outofbounds = true; idx = threadIdx.x;}

			idx_type pos = idx2pos3D(idx, nx, nslice, n_even);
			int z = pos/nslice;
			int y = (pos-z*nslice)/nx;
			int x = pos-z*nslice-y*nx;
			/////////////////////////////////////////////

			//Create Gaussian kernel
			///////////////////////////////////////////////////
			int fsize = (int) (3*sigma);
			float kernelsum = 0.0f;
			float valuesum = 0.0f;
			//////////////////////////////////////////////////

			for(int yi=-fsize; yi<=fsize; yi++)
			{
				int y0 = y+yi;

				//reflective boundaries
				if (y0 < 0) y0 = -y0;
				else if (y0 >= ny) y0 = 2*ny-y0-2;

				float kernel_val = expf(-(yi*yi)/(sigma*sigma*2));
				kernelsum += kernel_val;

				idx_type idx0 = xyz2idx(x, y0, z, nx, nslice, n_even);

				__syncthreads();
				if (y0 < 0 || y0 >= ny) continue;
				valuesum += kernel_val*input[idx0];
			}

			if(!outofbounds)
				output[idx] = valuesum/kernelsum;

			return;
		}
		__global__ void gaussianfilter3D_z(optflow_type *input, optflow_type *output)
		{
			//acquire constants
			/////////////////////////////////////////////
			float sigma = gpu_const::filter_sigma_c;

			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			idx_type n_even = nstack-(nstack/2);

			bool outofbounds = false;
			idx_type idx = (blockIdx.x*blockDim.x+threadIdx.x);
			if (idx >= nstack) {outofbounds = true; idx = threadIdx.x;}

			idx_type pos = idx2pos3D(idx, nx, nslice, n_even);
			int z = pos/nslice;
			int y = (pos-z*nslice)/nx;
			int x = pos-z*nslice-y*nx;
			/////////////////////////////////////////////

			//Create Gaussian kernel
			///////////////////////////////////////////////////
			int fsize = (int) (3*sigma);
			float kernelsum = 0.0f;
			float valuesum = 0.0f;
			//////////////////////////////////////////////////

			for(int zi=-fsize; zi<=fsize; zi++)
			{
				int z0 = z+zi;

				//reflective boundaries
				if (z0 < 0) z0 = -z0;
				else if (z0 >= nz) z0 = 2*nz-z0-2;

				float kernel_val = expf(-(zi*zi)/(sigma*sigma*2));
				kernelsum += kernel_val;

				idx_type idx0 = xyz2idx(x, y, z0, nx, nslice, n_even);

				__syncthreads();
				if (z0 < 0 || z0 >= nz) continue;
				valuesum += kernel_val*input[idx0];
			}

			if(!outofbounds)
				output[idx] = valuesum/kernelsum;

			return;
		}

		__global__ void calculate_sorUpdate(int iter, img_type *frame0, img_type *warped1, optflow_type *phi, optflow_type *psi, optflow_type *u, optflow_type *du, optflow_type *confidencemap)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			mathtype_solver epsilon_psi_squared = gpu_const::epsilon_psi_squared_c;
			mathtype_solver hx = gpu_const::hx_c;
			mathtype_solver hy = gpu_const::hy_c;
			mathtype_solver hz = gpu_const::hz_c;
			mathtype_solver alpha = gpu_const::alpha_c;
			mathtype_solver omega = gpu_const::omega_c;

			bool precalculated_psi = gpu_const::precalculated_psi_c;
			bool decoupled_smoothness = gpu_const::decoupled_smoothness_c;
			int slip_depth = gpu_const::slip_depth_c;

			float minIntensity = gpu_const::lowerIntensityCutoff_c;
			float maxIntensity = gpu_const::upperIntensityCutoff_c;

			mathtype_solver alphax = alpha/(hx*hx);
			mathtype_solver alphay = alpha/(hy*hy);
			mathtype_solver alphaz = alpha/(hz*hz);

			int spatiotemporalderivative_id = gpu_const::spatiotemporalderivative_id_c;
			bool use_confidencemap = gpu_const::use_confidencemap_c;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			idx_type nstack2 = 2*nstack;
			idx_type n_even = nstack-(nstack/2);

			//Adjust for even/odd updates in 2D
			///////////////////////////////////
			bool outofbounds = false;
			idx_type idx = (blockIdx.x*blockDim.x+threadIdx.x);

			if ((iter%2) == 0 && idx >= n_even) {outofbounds = true; idx = threadIdx.x;}
			else if ((iter%2) != 0)
			{
				idx += n_even;
				if (idx >= nstack) {outofbounds = true; idx = threadIdx.x+n_even;}
			}

			idx_type pos = idx2pos3D(idx, nx, nslice, n_even);
			int z = pos/nslice;
			int y = (pos-z*nslice)/nx;
			int x = pos-z*nslice-y*nx;
			///////////////////////////////////

			mathtype_solver confidence = 1.0f;
			mathtype_solver psi0 = 0.0f;
			mathtype_solver normalizer_x1 = 0.125f/hx;
			mathtype_solver normalizer_y1 = 0.125f/hy;
			mathtype_solver normalizer_z1 = 0.125f/hz;
			mathtype_solver normalizer_x2 = 0.25f/hx;
			mathtype_solver normalizer_y2 = 0.25f/hy;
			mathtype_solver normalizer_z2 = 0.25f/hz;
			/////////////////////////////////////////////

			//Define the neighbourhood
			/////////////////////////////////////////////
			int zp = z+1;
			int zn = z-1;
			int yp = y+1;
			int yn = y-1;
			int xp = x+1;
			int xn = x-1;

			mathtype_solver xp_active = 1.0f;
			mathtype_solver xn_active = 1.0f;
			mathtype_solver yp_active = 1.0f;
			mathtype_solver yn_active = 1.0f;
			mathtype_solver zp_active = 1.0f;
			mathtype_solver zn_active = 1.0f;

			bool boundary_voxel = false;
			if (xp == nx) {xp_active = 0.0f; xp = x; boundary_voxel = true;}
			else if (xn < 0) {xn_active = 0.0f; xn = x; boundary_voxel = true;}
			if (yp == ny) {yp_active = 0.0f; yp = y; boundary_voxel = true;}
			else if (yn < 0) {yn_active = 0.0f; yn = y; boundary_voxel = true;}
			if (zp == nz) {zp_active = 0.0f; zp = z; boundary_voxel = true;}
			else if (zn < 0) {zn_active = 0.0f; zn = z; boundary_voxel = true;}

			idx_type npos0 = xyz2idx(xp, y, z, nx, nslice, n_even);
			idx_type npos1 = xyz2idx(xn, y, z, nx, nslice, n_even);
			idx_type npos2 = xyz2idx(x, yp, z, nx, nslice, n_even);
			idx_type npos3 = xyz2idx(x, yn, z, nx, nslice, n_even);
			idx_type npos4 = xyz2idx(x, y, zp, nx, nslice, n_even);
			idx_type npos5 = xyz2idx(x, y, zn, nx, nslice, n_even);

			mathtype_solver phi_neighbour[18] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
			mathtype_solver du_neighbour[6]  = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
			mathtype_solver dv_neighbour[6]  = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
			mathtype_solver dw_neighbour[6]  = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
			/////////////////////////////////////////////

			//Switch to reflective boundary conditions
			/////////////////////////////////////////////
			zp = z+1;
			zn = z-1;
			yp = y+1;
			yn = y-1;
			xp = x+1;
			xn = x-1;

			if (zp == nz) zp -= 2;
			else if (zn == -1) zn = 1;
			if (yp == ny) yp -= 2;
			else if (yn == -1) yn = 1;
			if (xp == nx) xp -= 2;
			else if (xn == -1) xn = 1;
			/////////////////////////////////////////////

			/////////////////////////////////////////////
			__syncthreads();
			mathtype_solver phi0 = phi[idx];
			mathtype_solver u0 = u[idx];
			mathtype_solver v0 = u[idx+nstack];
			mathtype_solver w0 = u[idx+nstack2];
			mathtype_solver du0 = du[idx];
			mathtype_solver dv0 = du[idx+nstack];
			mathtype_solver dw0 = du[idx+nstack2];
			mathtype_solver frame0_val = frame0[idx];

			if (use_confidencemap) confidence = confidencemap[idx];
			if (precalculated_psi) psi0 = psi[idx];
			/////////////////////////////////////////////

			//Read in neighbours with 0-boundaries
			/////////////////////////////////////////////
			phi_neighbour[0] = xp_active*0.5f*(phi[npos0] + phi0);
			du_neighbour[0]  = u[npos0] + du[npos0] - u0;
			dv_neighbour[0]  = u[npos0 + nstack] + du[npos0 + nstack] - v0;
			dw_neighbour[0]  = u[npos0 + nstack2] + du[npos0 + nstack2] - w0;

			phi_neighbour[1] = xn_active*0.5f*(phi[npos1] + phi0);
			du_neighbour[1]  = u[npos1] + du[npos1] - u0;
			dv_neighbour[1]  = u[npos1 + nstack] + du[npos1 + nstack] - v0;
			dw_neighbour[1]  = u[npos1 + nstack2] + du[npos1 + nstack2] - w0;

			phi_neighbour[2] = yp_active*0.5f*(phi[npos2] + phi0);
			du_neighbour[2]  = u[npos2] + du[npos2] - u0;
			dv_neighbour[2]  = u[npos2 + nstack] + du[npos2 + nstack] - v0;
			dw_neighbour[2]  = u[npos2 + nstack2] + du[npos2 + nstack2] - w0;

			phi_neighbour[3] = yn_active*0.5f*(phi[npos3] + phi0);
			du_neighbour[3]  = u[npos3] + du[npos3] - u0;
			dv_neighbour[3]  = u[npos3 + nstack] + du[npos3 + nstack] - v0;
			dw_neighbour[3]  = u[npos3 + nstack2] + du[npos3 + nstack2] - w0;

			phi_neighbour[4] = zp_active*0.5f*(phi[npos4] + phi0);
			du_neighbour[4]  = u[npos4] + du[npos4] - u0;
			dv_neighbour[4]  = u[npos4 + nstack] + du[npos4 + nstack] - v0;
			dw_neighbour[4]  = u[npos4 + nstack2] + du[npos4 + nstack2] - w0;

			phi_neighbour[5] = zn_active*0.5f*(phi[npos5] + phi0);
			du_neighbour[5]  = u[npos5] + du[npos5] - u0;
			dv_neighbour[5]  = u[npos5 + nstack] + du[npos5 + nstack] - v0;
			dw_neighbour[5]  = u[npos5 + nstack2] + du[npos5 + nstack2] - w0;

			if(decoupled_smoothness)
			{
				mathtype_solver phi1 = phi[nstack+idx];
				mathtype_solver phi2 = phi[nstack2+idx];
				phi_neighbour[6] = xp_active*0.5f*(phi[nstack+npos0] + phi1);
				phi_neighbour[7] = xn_active*0.5f*(phi[nstack+npos1] + phi1);
				phi_neighbour[8] = yp_active*0.5f*(phi[nstack+npos2] + phi1);
				phi_neighbour[9] = yn_active*0.5f*(phi[nstack+npos3] + phi1);
				phi_neighbour[10]= zp_active*0.5f*(phi[nstack+npos4] + phi1);
				phi_neighbour[11]= zn_active*0.5f*(phi[nstack+npos5] + phi1);
				phi_neighbour[12] = xp_active*0.5f*(phi[nstack2+npos0] + phi2);
				phi_neighbour[13] = xn_active*0.5f*(phi[nstack2+npos1] + phi2);
				phi_neighbour[14] = yp_active*0.5f*(phi[nstack2+npos2] + phi2);
				phi_neighbour[15] = yn_active*0.5f*(phi[nstack2+npos3] + phi2);
				phi_neighbour[16]= zp_active*0.5f*(phi[nstack2+npos4] + phi2);
				phi_neighbour[17]= zn_active*0.5f*(phi[nstack2+npos5] + phi2);
			}
			/////////////////////////////////////////////

			mathtype_solver Idx, Idy, Idz, Idt;

			//Calculate spatiotemporal derivatives on the fly
			/////////////////////////////////////////////
			if (spatiotemporalderivative_id < 0){
				Idx = psi[idx+nstack];
				Idy = psi[idx+(2*nstack)];
				Idz = psi[idx+(3*nstack)];
				Idt = psi[idx+(4*nstack)];
			}
			else if (spatiotemporalderivative_id == 3){
				//Fourth Order Finite Difference
				//////////////////////////////////////////////////////////////////////////////////////////

				int yp2 = y+2; int yn2 = y-2; int xp2 = x+2; int xn2 = x-2; int zp2 = z+2; int zn2 = z-2;
				if (yp2 >= ny) yp2 = 2*ny-yp2-2; if (yn2 < 0) yn2 = -yn2;
				if (xp2 >= nx) xp2 = 2*nx-xp2-2; if (xn2 < 0) xn2 = -xn2;
				if (zp2 >= nz) zp2 = 2*nz-zp2-2; if (zn2 < 0) zn2 = -zn2;

				idx_type idx_xn2 = xyz2idx(xn2, y,z, nx, nslice, n_even);
				idx_type idx_xn = xyz2idx(xn, y,z, nx, nslice, n_even);
				idx_type idx_xp = xyz2idx(xp, y,z, nx, nslice, n_even);
				idx_type idx_xp2 = xyz2idx(xp2, y,z, nx, nslice, n_even);
				idx_type idx_yn2 = xyz2idx(x, yn2,z, nx, nslice, n_even);
				idx_type idx_yn = xyz2idx(x, yn,z, nx, nslice, n_even);
				idx_type idx_yp = xyz2idx(x, yp,z, nx, nslice, n_even);
				idx_type idx_yp2 = xyz2idx(x, yp2,z, nx, nslice, n_even);
				idx_type idx_zn2 = xyz2idx(x, y,zn2, nx, nslice, n_even);
				idx_type idx_zn = xyz2idx(x, y,zn, nx, nslice, n_even);
				idx_type idx_zp = xyz2idx(x, y,zp, nx, nslice, n_even);
				idx_type idx_zp2 = xyz2idx(x, y,zp2, nx, nslice, n_even);
				__syncthreads();

				mathtype_solver val_xn2_a = frame0[idx_xn2];
				mathtype_solver val_xn_a = frame0[idx_xn];
				mathtype_solver val_xp_a = frame0[idx_xp];
				mathtype_solver val_xp2_a = frame0[idx_xp2];
				mathtype_solver val_yn2_a = frame0[idx_yn2];
				mathtype_solver val_yn_a = frame0[idx_yn];
				mathtype_solver val_yp_a = frame0[idx_yp];
				mathtype_solver val_yp2_a = frame0[idx_yp2];
				mathtype_solver val_zn2_a = frame0[idx_zn2];
				mathtype_solver val_zn_a = frame0[idx_zn];
				mathtype_solver val_zp_a = frame0[idx_zp];
				mathtype_solver val_zp2_a = frame0[idx_zp2];

				mathtype_solver val_xn2_b = warped1[idx_xn2];
				mathtype_solver val_xn_b = warped1[idx_xn];
				mathtype_solver val0b    = warped1[idx];
				mathtype_solver val_xp_b = warped1[idx_xp];
				mathtype_solver val_xp2_b = warped1[idx_xp2];
				mathtype_solver val_yn2_b = warped1[idx_yn2];
				mathtype_solver val_yn_b = warped1[idx_yn];
				mathtype_solver val_yp_b = warped1[idx_yp];
				mathtype_solver val_yp2_b = warped1[idx_yp2];
				mathtype_solver val_zn2_b = warped1[idx_zn2];
				mathtype_solver val_zn_b = warped1[idx_zn];
				mathtype_solver val_zp_b = warped1[idx_zp];
				mathtype_solver val_zp2_b = warped1[idx_zp2];

				Idx = normalizer_x1/6.f*((val_xn2_a-8.f*val_xn_a+8.f*val_xp_a-val_xp2_a)+(val_xn2_b-8.f*val_xn_b+8.f*val_xp_b-val_xp2_b));
				Idy = normalizer_y1/6.f*((val_yn2_a-8.f*val_yn_a+8.f*val_yp_a-val_yp2_a)+(val_yn2_b-8.f*val_yn_b+8.f*val_yp_b-val_yp2_b));
				Idz = normalizer_z1/6.f*((val_zn2_a-8.f*val_zn_a+8.f*val_zp_a-val_zp2_a)+(val_zn2_b-8.f*val_zn_b+8.f*val_zp_b-val_zp2_b));
				Idt = val0b-frame0_val;
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			else if (spatiotemporalderivative_id == 2){
				//Ershov: average of frame1 and frame2, central difference, dt: forward difference
				//////////////////////////////////////////////////////////////////////////////////////////
				idx_type idx_xn = xyz2idx(xn, y, z, nx, nslice, n_even);
				idx_type idx_xp = xyz2idx(xp, y, z, nx, nslice, n_even);
				idx_type idx_yn = xyz2idx(x, yn, z, nx, nslice, n_even);
				idx_type idx_yp = xyz2idx(x, yp, z, nx, nslice, n_even);
				idx_type idx_zn = xyz2idx(x, y, zn, nx, nslice, n_even);
				idx_type idx_zp = xyz2idx(x, y, zp, nx, nslice, n_even);

				__syncthreads();

				mathtype_solver val_xn_a = frame0[idx_xn];
				mathtype_solver val_xp_a = frame0[idx_xp];
				mathtype_solver val_yn_a = frame0[idx_yn];
				mathtype_solver val_yp_a = frame0[idx_yp];
				mathtype_solver val_zn_a = frame0[idx_zn];
				mathtype_solver val_zp_a = frame0[idx_zp];

				mathtype_solver val_xn_b = warped1[idx_xn];
				mathtype_solver val0b    = warped1[idx];
				mathtype_solver val_xp_b = warped1[idx_xp];
				mathtype_solver val_yn_b = warped1[idx_yn];
				mathtype_solver val_yp_b = warped1[idx_yp];
				mathtype_solver val_zn_b = warped1[idx_zn];
				mathtype_solver val_zp_b = warped1[idx_zp];

				Idx = normalizer_x2*((val_xp_a-val_xn_a)+(val_xp_b-val_xn_b));
				Idy = normalizer_y2*((val_yp_a-val_yn_a)+(val_yp_b-val_yn_b));
				Idz = normalizer_z2*((val_zp_a-val_zn_a)+(val_zp_b-val_zn_b));
				Idt = val0b-frame0_val;
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			else if (spatiotemporalderivative_id == 1){
				//Horn-Schunck: average of frame1 and frame2, dx-kernel := [-1,1; -1,1], dt: local average
				//////////////////////////////////////////////////////////////////////////////////////////
				idx_type idx100 = xyz2idx(xp, y, z, nx, nslice, n_even);
				idx_type idx010 = xyz2idx(x, yp, z, nx, nslice, n_even);
				idx_type idx110 = xyz2idx(xp, yp, z, nx, nslice, n_even);
				idx_type idx001 = xyz2idx(x, y, zp, nx, nslice, n_even);
				idx_type idx101 = xyz2idx(xp, y, zp, nx, nslice, n_even);
				idx_type idx011 = xyz2idx(x, yp, zp, nx, nslice, n_even);
				idx_type idx111 = xyz2idx(xp, yp, zp, nx, nslice, n_even);

				__syncthreads();

				mathtype_solver val100a = frame0[idx100];
				mathtype_solver val010a = frame0[idx010];
				mathtype_solver val110a = frame0[idx110];
				mathtype_solver val001a = frame0[idx001];
				mathtype_solver val101a = frame0[idx101];
				mathtype_solver val011a = frame0[idx011];
				mathtype_solver val111a = frame0[idx111];

				mathtype_solver val000b = warped1[idx];
				mathtype_solver val100b = warped1[idx100];
				mathtype_solver val010b = warped1[idx010];
				mathtype_solver val110b = warped1[idx110];
				mathtype_solver val001b = warped1[idx001];
				mathtype_solver val101b = warped1[idx101];
				mathtype_solver val011b = warped1[idx011];
				mathtype_solver val111b = warped1[idx111];

				Idx = normalizer_x1*((-frame0_val + val100a - val010a + val110a) + (-val001a + val101a - val011a + val111a)
								   + (-val000b + val100b - val010b + val110b) + (-val001b + val101b - val011b + val111b));
				Idy = normalizer_y1*((-frame0_val - val100a + val010a + val110a) + (-val001a - val101a + val011a + val111a)
								   + (-val000b - val100b + val010b + val110b) + (-val001b - val101b + val011b + val111b));
				Idz = normalizer_z1*((-frame0_val - val100a + val001a + val101a) + (-val010a - val110a + val011a + val111a)
								   + (-val000b - val100b + val001b + val101b) + (-val010b - val110b + val011b + val111b));
				Idt = 0.125f*((val000b+val100b+val010b+val110b)+(val001b+val101b+val011b+val111b)-(frame0_val+val100a+val010a+val110a)-(val001a+val101a+val011a+val111a));
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			/////////////////////////////////////////////

			//Intensity constancy:
			/////////////////////////////////////////////
			mathtype_solver J11 = Idx*Idx;
			mathtype_solver J22 = Idy*Idy;
			mathtype_solver J33 = Idz*Idz;
			mathtype_solver J12 = Idx * Idy;
			mathtype_solver J13 = Idx * Idz;
			mathtype_solver J23 = Idy * Idz;
			mathtype_solver J14 = Idx * Idt;
			mathtype_solver J24 = Idy * Idt;
			mathtype_solver J34 = Idz * Idt;
			/////////////////////////////////////////////

			//Calculating data term on the fly doesn't hurt much and saves memory
			//(doesn't work for local global approach)
			////////////////////////////////////////////////////////////////
			if(!precalculated_psi)
			{
				//assuming inner_iterations = 1
				psi0 = Idt;//+Idx*du0+Idy*dv0+Idz*dw0;
				psi0 *= psi0;
			}
			psi0 = 0.5f/sqrtf(psi0+epsilon_psi_squared);

			if(use_confidencemap) psi0 *= max(0.0f, min(1.0f, confidence));

			//deactivate data term for backgound:
			if (frame0_val < minIntensity || frame0_val > maxIntensity) psi0 = 0.0f;
			if (slip_depth > 0 && (x < slip_depth || x >= nx-slip_depth || y < slip_depth || y >= ny-slip_depth || z < slip_depth || z >= nz-slip_depth)) psi0 = 0.0f;
			////////////////////////////////////////////////////////////////

			//Calculate SOR update
			/////////////////////////////////////////////
			mathtype_solver sumH = alphax*(phi_neighbour[0]+phi_neighbour[1]) + alphay*(phi_neighbour[2]+phi_neighbour[3]) + alphaz*(phi_neighbour[4]+phi_neighbour[5]);
			mathtype_solver sumU = alphax*(phi_neighbour[0]*du_neighbour[0] + phi_neighbour[1]*du_neighbour[1])
								 + alphay*(phi_neighbour[2]*du_neighbour[2] + phi_neighbour[3]*du_neighbour[3])
								 + alphaz*(phi_neighbour[4]*du_neighbour[4] + phi_neighbour[5]*du_neighbour[5]);

			mathtype_solver sumH2 = sumH;
			mathtype_solver sumH3 = sumH;
			mathtype_solver sumV, sumW;

			if(!decoupled_smoothness){
				sumV = alphax*(phi_neighbour[0]*dv_neighbour[0] + phi_neighbour[1]*dv_neighbour[1])
					 + alphay*(phi_neighbour[2]*dv_neighbour[2] + phi_neighbour[3]*dv_neighbour[3])
					 + alphaz*(phi_neighbour[4]*dv_neighbour[4] + phi_neighbour[5]*dv_neighbour[5]);
				sumW = alphax*(phi_neighbour[0]*dw_neighbour[0] + phi_neighbour[1]*dw_neighbour[1])
					 + alphay*(phi_neighbour[2]*dw_neighbour[2] + phi_neighbour[3]*dw_neighbour[3])
					 + alphaz*(phi_neighbour[4]*dw_neighbour[4] + phi_neighbour[5]*dw_neighbour[5]);
			}
			else
			{
				sumV = alphax*(phi_neighbour[6]*dv_neighbour[0]  + phi_neighbour[7]*dv_neighbour[1])
					 + alphay*(phi_neighbour[8]*dv_neighbour[2]  + phi_neighbour[9]*dv_neighbour[3])
					 + alphaz*(phi_neighbour[10]*dv_neighbour[4] + phi_neighbour[11]*dv_neighbour[5]);
				sumW = alphax*(phi_neighbour[12]*dw_neighbour[0] + phi_neighbour[13]*dw_neighbour[1])
					 + alphay*(phi_neighbour[14]*dw_neighbour[2] + phi_neighbour[15]*dw_neighbour[3])
					 + alphaz*(phi_neighbour[16]*dw_neighbour[4] + phi_neighbour[17]*dw_neighbour[5]);
				sumH2 = alphax*(phi_neighbour[6]+phi_neighbour[7])   + alphay*(phi_neighbour[8]+phi_neighbour[9])   + alphaz*(phi_neighbour[10]+phi_neighbour[11]);
				sumH3 = alphax*(phi_neighbour[12]+phi_neighbour[13]) + alphay*(phi_neighbour[14]+phi_neighbour[15]) + alphaz*(phi_neighbour[16]+phi_neighbour[17]);
			}

			mathtype_solver next_du, next_dv, next_dw;

			//SOR-step unless Dirichlet boundary conditions
			////////////////////////////////////////////////////////
			if (boundary_voxel)
			{
				if (   (x == 0 && gpu_const::fixedDirichletBoundary_c[2] == 1) || (x == nx-1 && gpu_const::fixedDirichletBoundary_c[3] == 1)
					|| (y == 0 && gpu_const::fixedDirichletBoundary_c[0] == 1) || (y == ny-1 && gpu_const::fixedDirichletBoundary_c[1] == 1)
					|| (z == 0 && gpu_const::fixedDirichletBoundary_c[4] == 1) || (z == nz-1 && gpu_const::fixedDirichletBoundary_c[5] == 1))
				{
					next_du = 0.0f;
					next_dv = 0.0f;
					next_dw = 0.0f;
				}
				else
				{
					if ((x == 0 && gpu_const::zeroDirichletBoundary_c[2] == 1) || (x == nx-1 && gpu_const::zeroDirichletBoundary_c[3] == 1))
						next_du = 0.0f; //boundary condition set
					else
						next_du = (1.f-omega)*du0 + omega*(psi0 *(-J14 -(J12*dv0)     -(J13*dw0)) + sumU)/((psi0*J11) + sumH);

					if ((y == 0 && gpu_const::zeroDirichletBoundary_c[0] == 1) || (y == ny-1 && gpu_const::zeroDirichletBoundary_c[1] == 1))
						next_dv = 0.0f;
					else
						next_dv = (1.f-omega)*dv0 + omega*(psi0 *(-J24 -(J12*next_du) -(J23*dw0)) + sumV)/((psi0*J22) + sumH2);

					if ((z == 0 && gpu_const::zeroDirichletBoundary_c[4] == 1) || (z == nz-1 && gpu_const::zeroDirichletBoundary_c[5] == 1))
						next_dw = 0.0f;
					else
						next_dw = (1.f-omega)*dw0 + omega*(psi0 *(-J34 -(J13*next_du) -(J23*next_dv)) + sumW)/((psi0*J33) + sumH3);
				}
			}
			else
			{
				next_du = (1.f-omega)*du0 + omega*(psi0 *(-J14 -(J12*dv0)     -(J13*dw0)) + sumU)/((psi0*J11) + sumH);
				next_dv = (1.f-omega)*dv0 + omega*(psi0 *(-J24 -(J12*next_du) -(J23*dw0)) + sumV)/((psi0*J22) + sumH2);
				next_dw = (1.f-omega)*dw0 + omega*(psi0 *(-J34 -(J13*next_du) -(J23*next_dv)) + sumW)/((psi0*J33) + sumH3);
			}

			if (gpu_const::protect_overlap_c)
			{
				//extend the Dirichlet boundary inwards for mosaic processing
				int half_overlap = gpu_const::overlap_c/2;

				if (    (x < half_overlap && gpu_const::fixedDirichletBoundary_c[2] == 1) || (x >= nx-1-half_overlap && gpu_const::fixedDirichletBoundary_c[3] == 1)
					 || (y < half_overlap && gpu_const::fixedDirichletBoundary_c[0] == 1) || (y >= ny-1-half_overlap && gpu_const::fixedDirichletBoundary_c[1] == 1)
					 || (z < half_overlap && gpu_const::fixedDirichletBoundary_c[4] == 1) || (z >= nz-1-half_overlap && gpu_const::fixedDirichletBoundary_c[5] == 1))
				{
					next_du = 0.0f;
					next_dv = 0.0f;
					next_dw = 0.0f;
				}
			}
			////////////////////////////////////////////////////////

			/////////////////////////////////////////////
			__syncthreads();
			if(!outofbounds)
			{
				du[idx] = next_du;
				du[idx+nstack] = next_dv;
				du[idx+nstack2] = next_dw;
			}
			/////////////////////////////////////////////

			return;
		}

		__global__ void addsolution_warpFrame1_z(bool rewarp, optflow_type *intermediate_warp, img_type *frame0, img_type *frame1, optflow_type *u, optflow_type *du)
		{
			//No need to allocate additional memory. We can use the phi array.

			//acquire constants and position
			/////////////////////////////////////////////
			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;
			if (nz == 1) return;

			//int outOfBounds_id = gpu_const::outOfBounds_id_c; <- only replace mode allowed here
			int interpolation_id = gpu_const::warpInterpolation_id_c;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			idx_type nstack2 = 2*nstack;
			idx_type n_even = nstack-(nstack/2);

			bool outofbounds = false;
			idx_type idx = (blockIdx.x*blockDim.x+threadIdx.x);
			if (idx >= nstack) {outofbounds = true; idx = threadIdx.x;}

			idx_type pos = idx2pos3D(idx, nx, nslice, n_even);
			int z = pos/nslice;
			int y = (pos-z*nslice)/nx;
			int x = pos-z*nslice-y*nx;
			/////////////////////////////////////////////

			/////////////////////////////////////////////
			__syncthreads();
			mathtype_solver w0 = u[idx+nstack2];
			mathtype_solver dw0 = du[idx+nstack2];

			w0 += dw0;
			float z0 = z + w0;

			if(rewarp) z0 = z + dw0;

			img_type replace_val = 0.0f;
			bool moved_out = false;

			//out of bounds?
			if (z0 < 0 || z0 > (nz-1)) {
				moved_out = true;
				replace_val = frame0[idx];
				z0 = z;
			}

			int zf = floor(z0);
			int zc = ceil(z0);

			float wz = z0-zf;
			float value = 0.0f;
			/////////////////////////////////////////////


			idx_type idx001 = xyz2idx(x,y,zf,nx,nslice,n_even);
			idx_type idx002 = xyz2idx(x,y,zc,nx,nslice,n_even);

			/////////////////////////////////////////////
			if (interpolation_id == 1) //cubic interpolation
			{
				//extrapolate with zero-gradient
				int zf2 = max(0, zf-1);
				int zc2 = min(zc+1, nz-1);

				idx_type idx000 = xyz2idx(x,y,zf2,nx,nslice,n_even);
				idx_type idx003 = xyz2idx(x,y,zc2,nx,nslice,n_even);

				__syncthreads();
				img_type P000 = frame1[idx000];
				img_type P001 = frame1[idx001];
				img_type P002 = frame1[idx002];
				img_type P003 = frame1[idx003];

				value = interpolate_cubic(P000,P001,P002,P003,wz);
			}
			else //linear interpolation
			{
				__syncthreads();
				img_type P000 = frame1[idx001];
				img_type P001 = frame1[idx002];

				value = (P001-P000)*wz+P000;
			}

			if(moved_out) value = replace_val;
			/////////////////////////////////////////////

			/////////////////////////////////////////////
			__syncthreads();
			if(!outofbounds)
			{
				intermediate_warp[idx] = value;
				u[idx+nstack2] = w0;
				du[idx+nstack2] = 0.0f;
			}
			/////////////////////////////////////////////

			return;
		}
		__global__ void addsolution_warpFrame1_xy(bool rewarp, img_type *warped1, img_type *frame0, optflow_type *frame1, optflow_type *u, optflow_type *du, optflow_type *confidence){

			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			int outOfBounds_id = gpu_const::outOfBounds_id_c;
			int interpolation_id = gpu_const::warpInterpolation_id_c;
			bool use_confidencemap = gpu_const::use_confidencemap_c;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			idx_type n_even = nstack-(nstack/2);

			bool outofbounds = false;
			idx_type idx = (blockIdx.x*blockDim.x+threadIdx.x);
			if (idx >= nstack) {outofbounds = true; idx = threadIdx.x;}

			idx_type pos = idx2pos3D(idx, nx, nslice, n_even);
			int z = pos/nslice;
			int y = (pos-z*nslice)/nx;
			int x = pos-z*nslice-y*nx;
			/////////////////////////////////////////////

			/////////////////////////////////////////////
			__syncthreads();
			mathtype_solver u0  = u[idx];
			mathtype_solver v0  = u[idx+nstack];
			mathtype_solver w0 = 0.0f; //should already be warped
			mathtype_solver du0 = du[idx];
			mathtype_solver dv0 = du[idx+nstack];

			if(nz > 1)
				w0 = u[idx+2*nstack]; //for out of bounds checking

			u0 += du0;
			v0 += dv0;

			float x0 = x + u0;
			float y0 = y + v0;
			float z0 = z + w0;

			if(rewarp)
			{
				x0 = x + du0;
				y0 = y + dv0;
				z0 = z;
			}

			//out of bounds?
			////////////////////////
			img_type replace_val = 0.0f;
			bool moved_out = false;

			if (y0 < 0 || x0 < 0 || z0 < 0 || x0 > (nx-1) || y0 > (ny-1) || z0 > (nz-1))
			{
				moved_out = true;
				if (outOfBounds_id == 0) replace_val = frame0[idx];
				else replace_val = gpu_const::nanf_c;

				if (use_confidencemap) confidence[idx] = 0.0f;

				x0 = x; y0 = y; //z0 = z;
			}
			////////////////////////

			int xf = floor(x0);
			int xc = ceil(x0);
			int yf = floor(y0);
			int yc = ceil(y0);

			//extrapolate with zero-gradient
			int xf2 = max(0, xf-1);
			int xc2 = min(xc+1, nx-1);
			int yf2 = max(0, yf-1);
			int yc2 = min(yc+1, ny-1);

			float wx = x0-xf;
			float wy = y0-yf;

			img_type value = 0.0f;

			/////////////////////////////////////////////
			idx_type idx11 = xyz2idx(xf, yf, z, nx, nslice, n_even);
			idx_type idx21 = xyz2idx(xc, yf, z, nx, nslice, n_even);
			idx_type idx12 = xyz2idx(xf, yc, z, nx, nslice, n_even);
			idx_type idx22 = xyz2idx(xc, yc, z, nx, nslice, n_even);

			if (interpolation_id == 1)
			{
				idx_type idx10 = xyz2idx(xf, yf2, z, nx, nslice, n_even);
				idx_type idx20 = xyz2idx(xc, yf2, z, nx, nslice, n_even);
				idx_type idx01 = xyz2idx(xf2, yf, z, nx, nslice, n_even);
				idx_type idx31 = xyz2idx(xc2, yf, z, nx, nslice, n_even);
				idx_type idx02 = xyz2idx(xf2, yc, z, nx, nslice, n_even);
				idx_type idx32 = xyz2idx(xc2, yc, z, nx, nslice, n_even);
				idx_type idx13 = xyz2idx(xf, yc2, z, nx, nslice, n_even);
				idx_type idx23 = xyz2idx(xc, yc2, z, nx, nslice, n_even);

				__syncthreads();
				img_type P11 = frame1[idx11];
				img_type P21 = frame1[idx21];
				img_type P12 = frame1[idx12];
				img_type P22 = frame1[idx22];

				img_type P10 = frame1[idx10];
				img_type P20 = frame1[idx20];
				img_type P01 = frame1[idx01];
				img_type P31 = frame1[idx31];
				img_type P02 = frame1[idx02];
				img_type P32 = frame1[idx32];
				img_type P13 = frame1[idx13];
				img_type P23 = frame1[idx23];

				float gtu = gpu3d_rs::interpolate_cubic(P01,P11,P21,P31,wx);
				float gbu = gpu3d_rs::interpolate_cubic(P02,P12,P22,P32,wx);

				float glv = gpu3d_rs::interpolate_cubic(P10,P11,P12,P13,wy);
				float grv = gpu3d_rs::interpolate_cubic(P20,P21,P22,P23,wy);

				float sigma_lr = (1.f-wx)*glv + wx*grv;
				float sigma_bt = (1.f-wy)*gtu + wy*gbu;
				float corr_lrbt = P11*(1.f-wy)*(1.f-wx) + P12*wy*(1.f-wx) + P21*(1.f-wy)*wx + P22*wx*wy;

				value = sigma_lr+sigma_bt-corr_lrbt;
			}
			else
			{
				__syncthreads();

				img_type P11 = frame1[idx11];
				img_type P21 = frame1[idx21];
				img_type P12 = frame1[idx12];
				img_type P22 = frame1[idx22];

				float glv = (P12-P11)*wy+P11; //left
				float grv = (P22-P21)*wy+P21; //right
				float gtu = (P21-P11)*wx+P11; //top
				float gbu = (P22-P12)*wx+P12; //bottom

				float sigma_lr = (1.f-wx)*glv + wx*grv;
				float sigma_bt = (1.f-wy)*gtu + wy*gbu;
				float corr_lrbt = P11*(1.f-wy)*(1.f-wx) + P12*wy*(1.f-wx) + P21*(1.f-wy)*wx + P22*wx*wy;

				value = sigma_lr+sigma_bt-corr_lrbt;
			}

			if(moved_out) value = replace_val;

			__syncthreads();
			////////////////////////////
			if(!outofbounds)
			{
				warped1[idx] = value;
				u[idx] = u0;
				u[idx+nstack] = v0;
				du[idx] = 0.0f;
				du[idx+nstack] = 0.0f;
			}
			////////////////////////////

			return;
		}
		__global__ void addsolution(optflow_type *u, optflow_type *du)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;

			idx_type pos = (blockIdx.x*blockDim.x+threadIdx.x);
			if (pos >= nstack) {pos = threadIdx.x;}
			/////////////////////////////////////////////

			/////////////////////////////////////////////
			__syncthreads();
			mathtype_solver u0  = u[pos];
			mathtype_solver v0  = u[pos+nstack];
			mathtype_solver w0  = u[pos+(2*nstack)];
			mathtype_solver du0 = du[pos];
			mathtype_solver dv0 = du[pos+nstack];
			mathtype_solver dw0  = du[pos+(2*nstack)];

			u0 += du0;
			v0 += dv0;
			w0 += dw0;

			u[pos] = u0;
			u[pos+nstack] = v0;
			u[pos+(2*nstack)] = w0;
			du[pos] = 0.0f;
			du[pos+nstack] = 0.0f;
			du[pos+(2*nstack)] = 0.0f;
			////////////////////////////

			return;
		}
		__global__ void update_dataterm(img_type *frame0, img_type *warped1, optflow_type *du, optflow_type *psi)
		{
			//acquire constants and position
			/////////////////////////////////////////////
			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			int spatiotemporalderivative_id = gpu_const::spatiotemporalderivative_id_c;
			mathtype_solver hx = gpu_const::hx_c;
			mathtype_solver hy = gpu_const::hy_c;
			mathtype_solver hz = gpu_const::hz_c;

			mathtype_solver normalizer_x1 = 0.125f/hx;
			mathtype_solver normalizer_y1 = 0.125f/hy;
			mathtype_solver normalizer_z1 = 0.125f/hz;
			mathtype_solver normalizer_x2 = 0.25f/hx;
			mathtype_solver normalizer_y2 = 0.25f/hy;
			mathtype_solver normalizer_z2 = 0.25f/hz;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;
			idx_type n_even = nstack-(nstack/2);

			idx_type idx = (blockIdx.x*blockDim.x+threadIdx.x);
			if (idx >= nstack) idx = threadIdx.x;
			idx_type pos = idx2pos3D(idx, nx, nslice, n_even);

			int z = pos/nslice;
			int y = (pos-z*nslice)/nx;
			int x = pos-z*nslice-y*nx;

			int zp = z+1; int zn = z-1;
			int yp = y+1; int yn = y-1;
			int xp = x+1; int xn = x-1;

			//Reflective boundary conditions
			if (zp == nz) zp -= 2; if (zn < 0) zn = 1;
			if (yp == ny) yp -= 2; if (yn < 0) yn = 1;
			if (xp == nx) xp -= 2; if (xn < 0) xn = 1;

			__syncthreads();
			/////////////////////////////////////////////

			float Idx, Idy, Idz, Idt;

			mathtype_solver du0 = du[idx];
			mathtype_solver dv0 = du[idx+nstack];
			mathtype_solver dw0 = du[idx+(2*nstack)];
			mathtype_solver frame0_val = frame0[idx];

			//Precalculate spatiotemporal derivatives for local-global
			/////////////////////////////////////////////
			if (abs(spatiotemporalderivative_id) == 3){
				//Fourth Order Finite Difference
				//////////////////////////////////////////////////////////////////////////////////////////

				int yp2 = y+2; int yn2 = y-2; int xp2 = x+2; int xn2 = x-2; int zp2 = z+2; int zn2 = z-2;
				if (yp2 >= ny) yp2 = 2*ny-yp2-2; if (yn2 < 0) yn2 = -yn2;
				if (xp2 >= nx) xp2 = 2*nx-xp2-2; if (xn2 < 0) xn2 = -xn2;
				if (zp2 >= nz) zp2 = 2*nz-zp2-2; if (zn2 < 0) zn2 = -zn2;

				idx_type idx_xn2 = xyz2idx(xn2, y,z, nx, nslice, n_even);
				idx_type idx_xn = xyz2idx(xn, y,z, nx, nslice, n_even);
				idx_type idx_xp = xyz2idx(xp, y,z, nx, nslice, n_even);
				idx_type idx_xp2 = xyz2idx(xp2, y,z, nx, nslice, n_even);
				idx_type idx_yn2 = xyz2idx(x, yn2,z, nx, nslice, n_even);
				idx_type idx_yn = xyz2idx(x, yn,z, nx, nslice, n_even);
				idx_type idx_yp = xyz2idx(x, yp,z, nx, nslice, n_even);
				idx_type idx_yp2 = xyz2idx(x, yp2,z, nx, nslice, n_even);
				idx_type idx_zn2 = xyz2idx(x, y,zn2, nx, nslice, n_even);
				idx_type idx_zn = xyz2idx(x, y,zn, nx, nslice, n_even);
				idx_type idx_zp = xyz2idx(x, y,zp, nx, nslice, n_even);
				idx_type idx_zp2 = xyz2idx(x, y,zp2, nx, nslice, n_even);
				__syncthreads();

				mathtype_solver val_xn2_a = frame0[idx_xn2];
				mathtype_solver val_xn_a = frame0[idx_xn];
				mathtype_solver val_xp_a = frame0[idx_xp];
				mathtype_solver val_xp2_a = frame0[idx_xp2];
				mathtype_solver val_yn2_a = frame0[idx_yn2];
				mathtype_solver val_yn_a = frame0[idx_yn];
				mathtype_solver val_yp_a = frame0[idx_yp];
				mathtype_solver val_yp2_a = frame0[idx_yp2];
				mathtype_solver val_zn2_a = frame0[idx_zn2];
				mathtype_solver val_zn_a = frame0[idx_zn];
				mathtype_solver val_zp_a = frame0[idx_zp];
				mathtype_solver val_zp2_a = frame0[idx_zp2];

				mathtype_solver val_xn2_b = warped1[idx_xn2];
				mathtype_solver val_xn_b = warped1[idx_xn];
				mathtype_solver val0b    = warped1[idx];
				mathtype_solver val_xp_b = warped1[idx_xp];
				mathtype_solver val_xp2_b = warped1[idx_xp2];
				mathtype_solver val_yn2_b = warped1[idx_yn2];
				mathtype_solver val_yn_b = warped1[idx_yn];
				mathtype_solver val_yp_b = warped1[idx_yp];
				mathtype_solver val_yp2_b = warped1[idx_yp2];
				mathtype_solver val_zn2_b = warped1[idx_zn2];
				mathtype_solver val_zn_b = warped1[idx_zn];
				mathtype_solver val_zp_b = warped1[idx_zp];
				mathtype_solver val_zp2_b = warped1[idx_zp2];

				Idx = normalizer_x1/6.f*((val_xn2_a-8.f*val_xn_a+8.f*val_xp_a-val_xp2_a)+(val_xn2_b-8.f*val_xn_b+8.f*val_xp_b-val_xp2_b));
				Idy = normalizer_y1/6.f*((val_yn2_a-8.f*val_yn_a+8.f*val_yp_a-val_yp2_a)+(val_yn2_b-8.f*val_yn_b+8.f*val_yp_b-val_yp2_b));
				Idz = normalizer_z1/6.f*((val_zn2_a-8.f*val_zn_a+8.f*val_zp_a-val_zp2_a)+(val_zn2_b-8.f*val_zn_b+8.f*val_zp_b-val_zp2_b));
				Idt = val0b-frame0_val;
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			else if (abs(spatiotemporalderivative_id) == 2){
				//Ershov: average of frame1 and frame2, central difference, dt: forward difference
				//////////////////////////////////////////////////////////////////////////////////////////
				idx_type idx_xn = xyz2idx(xn, y, z, nx, nslice, n_even);
				idx_type idx_xp = xyz2idx(xp, y, z, nx, nslice, n_even);
				idx_type idx_yn = xyz2idx(x, yn, z, nx, nslice, n_even);
				idx_type idx_yp = xyz2idx(x, yp, z, nx, nslice, n_even);
				idx_type idx_zn = xyz2idx(x, y, zn, nx, nslice, n_even);
				idx_type idx_zp = xyz2idx(x, y, zp, nx, nslice, n_even);

				__syncthreads();

				mathtype_solver val_xn_a = frame0[idx_xn];
				mathtype_solver val_xp_a = frame0[idx_xp];
				mathtype_solver val_yn_a = frame0[idx_yn];
				mathtype_solver val_yp_a = frame0[idx_yp];
				mathtype_solver val_zn_a = frame0[idx_zn];
				mathtype_solver val_zp_a = frame0[idx_zp];

				mathtype_solver val_xn_b = warped1[idx_xn];
				mathtype_solver val0b    = warped1[idx];
				mathtype_solver val_xp_b = warped1[idx_xp];
				mathtype_solver val_yn_b = warped1[idx_yn];
				mathtype_solver val_yp_b = warped1[idx_yp];
				mathtype_solver val_zn_b = warped1[idx_zn];
				mathtype_solver val_zp_b = warped1[idx_zp];

				Idx = normalizer_x2*((val_xp_a-val_xn_a)+(val_xp_b-val_xn_b));
				Idy = normalizer_y2*((val_yp_a-val_yn_a)+(val_yp_b-val_yn_b));
				Idz = normalizer_z2*((val_zp_a-val_zn_a)+(val_zp_b-val_zn_b));
				Idt = val0b-frame0_val;
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			else if (abs(spatiotemporalderivative_id) == 1){
				//Horn-Schunck: average of frame1 and frame2, dx-kernel := [-1,1; -1,1], dt: local average
				//////////////////////////////////////////////////////////////////////////////////////////
				idx_type idx100 = xyz2idx(xp, y, z, nx, nslice, n_even);
				idx_type idx010 = xyz2idx(x, yp, z, nx, nslice, n_even);
				idx_type idx110 = xyz2idx(xp, yp, z, nx, nslice, n_even);
				idx_type idx001 = xyz2idx(x, y, zp, nx, nslice, n_even);
				idx_type idx101 = xyz2idx(xp, y, zp, nx, nslice, n_even);
				idx_type idx011 = xyz2idx(x, yp, zp, nx, nslice, n_even);
				idx_type idx111 = xyz2idx(xp, yp, zp, nx, nslice, n_even);

				__syncthreads();

				mathtype_solver val100a = frame0[idx100];
				mathtype_solver val010a = frame0[idx010];
				mathtype_solver val110a = frame0[idx110];
				mathtype_solver val001a = frame0[idx001];
				mathtype_solver val101a = frame0[idx101];
				mathtype_solver val011a = frame0[idx011];
				mathtype_solver val111a = frame0[idx111];

				mathtype_solver val000b = warped1[idx];
				mathtype_solver val100b = warped1[idx100];
				mathtype_solver val010b = warped1[idx010];
				mathtype_solver val110b = warped1[idx110];
				mathtype_solver val001b = warped1[idx001];
				mathtype_solver val101b = warped1[idx101];
				mathtype_solver val011b = warped1[idx011];
				mathtype_solver val111b = warped1[idx111];

				Idx = normalizer_x1*((-frame0_val + val100a - val010a + val110a) + (-val001a + val101a - val011a + val111a)
								   + (-val000b + val100b - val010b + val110b) + (-val001b + val101b - val011b + val111b));
				Idy = normalizer_y1*((-frame0_val - val100a + val010a + val110a) + (-val001a - val101a + val011a + val111a)
								   + (-val000b - val100b + val010b + val110b) + (-val001b - val101b + val011b + val111b));
				Idz = normalizer_z1*((-frame0_val - val100a + val001a + val101a) + (-val010a - val110a + val011a + val111a)
								   + (-val000b - val100b + val001b + val101b) + (-val010b - val110b + val011b + val111b));
				Idt = 0.125f*((val000b+val100b+val010b+val110b)+(val001b+val101b+val011b+val111b)-(frame0_val+val100a+val010a+val110a)-(val001a+val101a+val011a+val111a));
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			/////////////////////////////////////////////

			mathtype_solver psi0 = Idt+Idx*du0+Idy*dv0+Idz*dw0;
			psi0 *= psi0;

			__syncthreads();
			psi[idx] = psi0;

			if (spatiotemporalderivative_id < 0)
			{
				psi[nstack+idx] = Idx;
				psi[(2*nstack)+idx] = Idy;
				psi[(3*nstack)+idx] = Idz;
				psi[(4*nstack)+idx] = Idt;
			}

			return;
		}

		__global__ void zeroinitialize(optflow_type *u, optflow_type *du, optflow_type *confidence)
		{
			int ndim = gpu_const::ndim_c;
			idx_type nstack = gpu_const::nstack_c;
			bool use_confidencemap = gpu_const::use_confidencemap_c;

			idx_type pos = (blockIdx.x*blockDim.x+threadIdx.x);
			if (pos >= nstack) pos = threadIdx.x;
			__syncthreads();

			if(ndim > 2)
			{
				u[pos] = 0.0f;
				u[pos+nstack] = 0.0f;
				u[pos+2*nstack] = 0.0f;
				du[pos] = 0.0f;
				du[pos+nstack] = 0.0f;
				du[pos+2*nstack] = 0.0f;
			}
			else
			{
				u[pos] = 0.0f;
				u[pos+nstack] = 0.0f;
				du[pos] = 0.0f;
				du[pos+nstack] = 0.0f;
			}
			if (use_confidencemap)
			{
				confidence[pos] = 1.0f;
			}

			return;
		}
	}

	void OptFlow_GPU3D_Reshape::reshape_on_host(float *input, float *output, int shape[3])
	{
		int nx = shape[0];
		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = nslice*shape[2];

		idx_type n_odd = nstack/2;
		idx_type n_even = nstack-n_odd;

		#pragma omp parallel for
		for (idx_type idx = 0; idx < nstack; idx++)
		{
			idx_type pos;

			if(idx < n_even)
			{
				pos = 2*idx;
				int z = pos/nslice;
				int y = (pos-z*nslice)/nx;

					 if ((nx%2) == 0 && (y%2) != 0 && (z%2) == 0) pos++;
				else if ((nx%2) == 0 && (y%2) == 0 && (z%2) != 0) pos++;
				else if ((nx%2) != 0 && (nslice%2) == 0 && (z%2) != 0) pos++;
			}
			else
			{
				pos = (idx-n_even)*2;
				int z = pos/nslice;
				int y = (pos-z*nslice)/nx;

					 if((nx%2) != 0 && (z%2) == 0) pos++;
				else if((nslice%2) != 0 && (z%2) != 0) pos++;
				else if((nx%2) == 0 && (y%2) == 0 && (z%2) == 0) pos++;
				else if((nx%2) == 0 && (y%2) != 0 && (z%2) != 0) pos++;
			}

			if(pos < nstack)
				output[idx] = input[pos];
		}

		return;
	}

	int OptFlow_GPU3D_Reshape::configure_device(int maxshape[3], ProtocolParameters *params){

		deviceID = params->gpu.deviceID;
		hipSetDevice(deviceID);

		idx_type ndim = 3;
		bool use_confidencemap = params->confidence.use_confidencemap;
		idx_type nstack = maxshape[0]*maxshape[1];
		nstack *= maxshape[2];

		mathtype_solver epsilon_phi_squared = params->smoothness.epsilon_phi;
		epsilon_phi_squared *= epsilon_phi_squared;
		mathtype_solver epsilon_psi_squared = params->solver.epsilon_psi;
		epsilon_psi_squared *= epsilon_psi_squared;
		float nanf = std::numeric_limits<float>::quiet_NaN();

		int outOfBounds_id = 0;
		int warp_interpolation_id = 0;
		int spatiotemporalderivative_id = 0;

			 if (params->warp.outOfBounds_mode == "replace") outOfBounds_id = 0;
		else if (params->warp.outOfBounds_mode == "NaN") outOfBounds_id = 1;
		else std::cout << "Warning! Unknown outOfBounds_mode!" << std::endl;

			 if (params->warp.interpolation_mode == "cubic") warp_interpolation_id = 1;
		else if (params->warp.interpolation_mode == "linear") warp_interpolation_id = 0;
		else std::cout << "Warning! Unknow warp interpolation mode!" << std::endl;

			 if (params->solver.spatiotemporalDerivative_type == "HornSchunck") spatiotemporalderivative_id = 1;
		else if (params->solver.spatiotemporalDerivative_type == "Ershov") spatiotemporalderivative_id = 2;
		else if (params->solver.spatiotemporalDerivative_type == "centraldifference") spatiotemporalderivative_id = 2;
		else if (params->solver.spatiotemporalDerivative_type == "Barron") spatiotemporalderivative_id = 3;
		else {std::cout << "Warning! Unknown spatiotemporal derivative type!" << std::endl;}

		//identify precalculated spatiotemporal derivatives:
		if (params->solver.precalculate_derivatives) spatiotemporalderivative_id *= -1;

		bool anisotropic_smoothness = params->smoothness.anisotropic_smoothness;
		bool decoupled_smoothness = params->smoothness.decoupled_smoothness;
		bool adaptive_smoothness = params->smoothness.adaptive_smoothness;
		bool complementary_smoothness = params->smoothness.complementary_smoothness;

		int slip_depth = params->confidence.slip_depth;

		//check memory requirements
		////////////////////////////////////////////////////
		size_t free_byte, total_byte ;
		hipMemGetInfo( &free_byte, &total_byte ) ;

		int n_optflow = 7;
		int n_img = 2;

		double free_db = (double)free_byte ;
		double expected_usage = 7.*nstack *sizeof(optflow_type);
		expected_usage += 2.*nstack *sizeof(img_type);
		if(params->confidence.use_confidencemap) {expected_usage += nstack *sizeof(optflow_type); n_optflow++;}
		if (params->solver.precalculate_derivatives) {expected_usage += (5*nstack)*sizeof(optflow_type); n_optflow+=5;}
		else if(params->solver.precalculate_psi) {expected_usage += nstack *sizeof(optflow_type); n_optflow++;}
		if(params->warp.rewarp_frame1 == false) {expected_usage += nstack *sizeof(img_type); n_img++;}
		if(params->smoothness.decoupled_smoothness) {expected_usage += (2*nstack) *sizeof(optflow_type); n_optflow+=2;}
		if(params->smoothness.adaptive_smoothness)  {expected_usage += (2*nstack) *sizeof(optflow_type); n_optflow+=2;}
		if(params->solver.precalculate_derivatives && params->special.localglobal_dataterm && !params->smoothness.decoupled_smoothness) {expected_usage += (4*nstack) *sizeof(optflow_type); n_optflow+=4;}
		else if (params->solver.precalculate_derivatives && params->special.localglobal_dataterm && params->smoothness.decoupled_smoothness) {expected_usage += (2*nstack) *sizeof(optflow_type); n_optflow+=2;}

		if (params->mosaicing.mosaic_decomposition && params->mosaicing.max_nstack == -1)
		{
			params->mosaicing.max_nstack = (free_db-(params->mosaicing.memory_buffer*1024*1024))/(n_optflow*sizeof(optflow_type)+n_img*sizeof(img_type));

			if (nstack > params->mosaicing.max_nstack)
			{
				//set nstack to available memory
				expected_usage = expected_usage/nstack*params->mosaicing.max_nstack;
				nstack = params->mosaicing.max_nstack;
				std::cout << "\033[1;31mmax allowed nstack: " << nstack << "\033[0m" << std::endl;
				std::cout << "\033[1;32mGPU memory: " << round(expected_usage/(1024.*1024.)) << " MB out of " << round(free_db/(1024.*1024.)) << " MB\033[0m" << std::endl;
			}
			else
			{
				//deactivate
				params->mosaicing.mosaic_decomposition = false;
				std::cout << "\033[1;32mGPU memory: " << round(expected_usage/(1024.*1024.)) << " MB out of " << round(free_db/(1024.*1024.)) << " MB\033[0m" << std::endl;
			}
		}
		else
		{
			if (expected_usage > free_db){std::cout << "\033[1;31mError! Expected to run out of GPU memory!\033[0m" << std::endl;return 2;}
			else std::cout << "\033[1;32mGPU memory: " << round(expected_usage/(1024.*1024.)) << " MB out of " << round(free_db/(1024.*1024.)) << " MB\033[0m" << std::endl;
		}
		////////////////////////////////////////////////////

		if (params->mosaicing.mosaic_decomposition && params->mosaicing.sequential_approximation == false && params->gpu.n_gpus == 1)
			params->warp.rewarp_frame1 = true; //no reason to keep frame1 in GPU memory (with single GPU)

		//allocate memory and set constant memory
		////////////////////////////////////////////////////
		(optflow_type*) hipMalloc((void**)&u, (ndim*nstack)*sizeof(*u));
		(optflow_type*) hipMalloc((void**)&du, (ndim*nstack)*sizeof(*du));
		if(params->solver.precalculate_derivatives && params->special.localglobal_dataterm) hipMalloc((void**)&phi, (5*nstack)*sizeof(*phi));
		else if(params->smoothness.decoupled_smoothness) (optflow_type*) hipMalloc((void**)&phi, (3*nstack)*sizeof(*phi));
		else (optflow_type*) hipMalloc((void**)&phi, nstack*sizeof(*phi));
		//confidence map or background mask:
		if(params->confidence.use_confidencemap) (optflow_type*) hipMalloc((void**)&confidence, nstack*sizeof(*confidence));
		else (optflow_type*) hipMalloc((void**)&confidence, 0);
		if (params->solver.precalculate_derivatives) (optflow_type*) hipMalloc((void**)&psi, (5*nstack)*sizeof(*psi));
		else if(params->solver.precalculate_psi) (optflow_type*) hipMalloc((void**)&psi, nstack*sizeof(*psi));
		else (optflow_type*) hipMalloc((void**)&psi, 0);
		if(params->smoothness.adaptive_smoothness) (optflow_type*) hipMalloc((void**)&adaptivity, (2*nstack)*sizeof(*adaptivity));
		else (optflow_type*) hipMalloc((void**)&adaptivity, 0);

		//using an extra copy to warp from source (rewarp would save a copy)
		(img_type*) hipMalloc((void**)&warped1, nstack*sizeof(*warped1));
		(img_type*) hipMalloc((void**)&dev_frame0, nstack*sizeof(*dev_frame0));
		if (params->warp.rewarp_frame1 == false) (img_type*) hipMalloc((void**)&dev_frame1, nstack*sizeof(*dev_frame1));
		else (img_type*) hipMalloc((void**)&dev_frame1, 0);

		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::ndim_c), &ndim, sizeof(gpu_const::ndim_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nstack_c), &nstack, sizeof(gpu_const::nstack_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::use_confidencemap_c), &use_confidencemap, sizeof(gpu_const::use_confidencemap_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::epsilon_phi_squared_c), &epsilon_phi_squared, sizeof(gpu_const::epsilon_phi_squared_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::epsilon_psi_squared_c), &epsilon_psi_squared, sizeof(gpu_const::epsilon_psi_squared_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::outOfBounds_id_c), &outOfBounds_id, sizeof(gpu_const::outOfBounds_id_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::warpInterpolation_id_c), &warp_interpolation_id, sizeof(gpu_const::warpInterpolation_id_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::spatiotemporalderivative_id_c), &spatiotemporalderivative_id, sizeof(gpu_const::spatiotemporalderivative_id_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nanf_c), &nanf, sizeof(gpu_const::nanf_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::anisotropic_smoothness_c), &anisotropic_smoothness, sizeof(gpu_const::anisotropic_smoothness_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::decoupled_smoothness_c), &decoupled_smoothness, sizeof(gpu_const::decoupled_smoothness_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::adaptive_smoothness_c), &adaptive_smoothness, sizeof(gpu_const::adaptive_smoothness_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::complementary_smoothness_c), &complementary_smoothness, sizeof(gpu_const::complementary_smoothness_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::slip_depth_c), &slip_depth, sizeof(gpu_const::slip_depth_c));
		hipDeviceSynchronize();
		////////////////////////////////////////////////////

		//Initialize arrays
		////////////////////////////////////////////////////
		int threadsPerBlock(params->gpu.threadsPerBlock);
		int blocksPerGrid = (nstack + threadsPerBlock - 1) / (threadsPerBlock);

		gpu3d_rs::zeroinitialize<<<blocksPerGrid,threadsPerBlock>>>(u,du,confidence);
		hipDeviceSynchronize();
		////////////////////////////////////////////////////

		std::string error_string = (std::string) hipGetErrorString(hipGetLastError());
		if (error_string != "no error")
		{
			std::cout << "Device Variable Copying: " << error_string << std::endl;
			return 1;
		}

		return 0;
	}
	void OptFlow_GPU3D_Reshape::free_device(){
		hipSetDevice(deviceID);

		hipFree(u);
		hipFree(du);
		hipFree(phi);
		hipFree(psi);
		hipFree(confidence);
		hipFree(dev_frame0);
		hipFree(dev_frame1);
		hipFree(warped1);
		hipFree(adaptivity);
	}

	void OptFlow_GPU3D_Reshape::run_outeriterations(int level, img_type *frame0, img_type *frame1, int shape[3], ProtocolParameters *params, bool resumed_state, bool frames_set)
	{
		hipSetDevice(deviceID);

		//Set constant memory
		////////////////////////////////////////////////////////////////////////////////////////
		int nx = shape[0]; int ny = shape[1]; int nz = shape[2];
		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = shape[2]*nslice;
		idx_type n_odd = nstack/2;
		idx_type n_even = nstack-n_odd;

		int threadsPerBlock(params->gpu.threadsPerBlock);
		int blocksPerGrid = (nstack + threadsPerBlock - 1) / (threadsPerBlock);
		int blocksPerGrid2 = (n_even + threadsPerBlock -1) / (threadsPerBlock); //iterate over every second voxel

		idx_type asize1 = nstack*sizeof(*dev_frame0);

		mathtype_solver hx = params->scaling.hx;
		mathtype_solver hy = params->scaling.hy;
		mathtype_solver hz = params->scaling.hz;
		mathtype_solver alpha = params->alpha;
		mathtype_solver omega = params->solver.sor_omega;
		bool precalculate_psi = params->solver.precalculate_psi;
		float localglobal_sigma_data = params->special.localglobal_sigma_data;
		bool rewarp = params->warp.rewarp_frame1;
		bool use_confidencemap = params->confidence.use_confidencemap;
		bool protect_overlap = params->mosaicing.protect_overlap;
		int overlap = params->mosaicing.overlap;

		if (params->pyramid.alpha_scaling)
			alpha = alpha/pow(params->pyramid.scaling_factor, level);

		int smoothness_id = 0;

			 if (params->solver.flowDerivative_type == "Barron") smoothness_id = 0;
		else if (params->solver.flowDerivative_type == "centraldifference") smoothness_id = 1; //Ershov style
		else if (params->solver.flowDerivative_type == "forwarddifference") smoothness_id = 2; //Liu style
		else std::cout << "Warning! Unknown flowDerivative_type!" << std::endl;
		if (!resumed_state)
		{
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nx_c), &nx, sizeof(gpu_const::nx_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::ny_c), &ny, sizeof(gpu_const::ny_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nz_c), &nz, sizeof(gpu_const::nz_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nstack_c), &nstack, sizeof(gpu_const::nstack_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::hx_c), &hx, sizeof(gpu_const::hx_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::hy_c), &hy, sizeof(gpu_const::hy_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::hz_c), &hz, sizeof(gpu_const::hz_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::alpha_c), &alpha, sizeof(gpu_const::alpha_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::omega_c), &omega, sizeof(gpu_const::omega_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::zeroDirichletBoundary_c), &params->constraint.zeroDirichletBoundary,  6*sizeof(int), 0);
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::fixedDirichletBoundary_c), &params->constraint.fixedDirichletBoundary,  6*sizeof(int), 0);
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::lowerIntensityCutoff_c), &(params->constraint.intensityRange[0]), sizeof(gpu_const::lowerIntensityCutoff_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::upperIntensityCutoff_c), &(params->constraint.intensityRange[1]), sizeof(gpu_const::upperIntensityCutoff_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::use_confidencemap_c), &use_confidencemap, sizeof(gpu_const::use_confidencemap_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::precalculated_psi_c), &precalculate_psi, sizeof(gpu_const::precalculated_psi_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::filter_sigma_c), &localglobal_sigma_data, sizeof(gpu_const::filter_sigma_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::protect_overlap_c), &protect_overlap, sizeof(gpu_const::protect_overlap_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::overlap_c), &overlap, sizeof(gpu_const::overlap_c));

			//Reshape the image to even-odd-format and copy to device
			//////////////////////////////////////////
			if (!frames_set)
			{
				img_type *reshaped = (img_type*) malloc(nstack*sizeof(*reshaped));
				reshape_on_host(frame0, reshaped, shape);
				hipMemcpy(dev_frame0, reshaped, asize1, hipMemcpyHostToDevice);
				hipDeviceSynchronize();
				reshape_on_host(frame1, reshaped, shape);
				if(!rewarp) hipMemcpy(dev_frame1, reshaped, asize1, hipMemcpyHostToDevice);
				else hipMemcpy(warped1, reshaped, asize1, hipMemcpyHostToDevice);
				hipDeviceSynchronize();
				free(reshaped);
			}
			//////////////////////////////////////////

			////////////////////////////////////////////////////////////////////////////////////////
			//initial warp for frame 1
			if(!rewarp){
				gpu3d_rs::addsolution_warpFrame1_z<<<blocksPerGrid,threadsPerBlock>>>(false, phi, dev_frame0, dev_frame1, u, du);
				hipDeviceSynchronize();
				gpu3d_rs::addsolution_warpFrame1_xy<<<blocksPerGrid,threadsPerBlock>>>(false, warped1, dev_frame0, phi, u, du, confidence);
				hipDeviceSynchronize();
			}
			else{
				gpu3d_rs::addsolution_warpFrame1_z<<<blocksPerGrid,threadsPerBlock>>>(false, phi, dev_frame0, warped1, u, du);
				hipDeviceSynchronize();
				gpu3d_rs::addsolution_warpFrame1_xy<<<blocksPerGrid,threadsPerBlock>>>(false, warped1, dev_frame0, phi, u, du, confidence);
				hipDeviceSynchronize();
			}
		}

		for (int i_outer = 0; i_outer < params->solver.outerIterations; i_outer++)
		{
			std::cout << "level " << level << " (" << nx << "," << ny << "," << nz << "): " << (i_outer+1) << " \r";
			std::cout.flush();

			for (int i_inner = 0; i_inner < params->solver.innerIterations; i_inner++)
			{
				if (params->special.localglobal_dataterm)
				{
					gpu3d_rs::update_dataterm<<<blocksPerGrid,threadsPerBlock>>>(dev_frame0, warped1, du, phi);
					hipDeviceSynchronize();

					//could be reduced to 4 in the next version without storing psi0
					int maxoffset = params->solver.precalculate_derivatives ? 5 : 1; //needs to be always 5 or it's not proper local-global since the Tensor needs to be blurred

					for (long long int offset = 0; offset < maxoffset*nstack; offset += nstack)
					{
					gpu3d_rs::gaussianfilter3D_x<<<blocksPerGrid,threadsPerBlock>>>(phi, psi);
					hipDeviceSynchronize();
					gpu3d_rs::gaussianfilter3D_y<<<blocksPerGrid,threadsPerBlock>>>(psi, phi);
					hipDeviceSynchronize();
					gpu3d_rs::gaussianfilter3D_z<<<blocksPerGrid,threadsPerBlock>>>(phi, psi);
					hipDeviceSynchronize();
					}
				}
				else if(precalculate_psi)
				{
					gpu3d_rs::update_dataterm<<<blocksPerGrid,threadsPerBlock>>>(dev_frame0, warped1, du, psi);
					//hipDeviceSynchronize();
				}

				//Calculate the smoothness term
				//////////////////////////////////////////////////////////////////////////////
				if      (smoothness_id == 0) gpu3d_rs::update_smoothnessterm_Barron<<<blocksPerGrid,threadsPerBlock>>>(u, du,  phi, adaptivity);
				else if (smoothness_id == 1) gpu3d_rs::update_smoothnessterm_centralDiff<<<blocksPerGrid,threadsPerBlock>>>(u, du,  phi, adaptivity);
				else if (smoothness_id == 2) gpu3d_rs::update_smoothnessterm_forwardDiff<<<blocksPerGrid,threadsPerBlock>>>(u, du,  phi, adaptivity);
				hipDeviceSynchronize();

				//////////////////////////////////////////////////////////////////////////////

				//SOR-Updates with psi calculated on the fly
				//////////////////////////////////////////////////////////////////////////////
				//switching between even and odd
				for (int i_sor = 0; i_sor < 2*params->solver.sorIterations; i_sor++)
				{
					gpu3d_rs::calculate_sorUpdate<<<blocksPerGrid2,threadsPerBlock>>>(i_sor, dev_frame0, warped1, phi, psi, u, du, confidence);
					//hipDeviceSynchronize();
				}
				//////////////////////////////////////////////////////////////////////////////
			}

			if(!rewarp){
				gpu3d_rs::addsolution_warpFrame1_z<<<blocksPerGrid,threadsPerBlock>>>(false, phi, dev_frame0, dev_frame1, u, du);
				hipDeviceSynchronize();
				gpu3d_rs::addsolution_warpFrame1_xy<<<blocksPerGrid,threadsPerBlock>>>(false, warped1, dev_frame0, phi, u, du, confidence);
				hipDeviceSynchronize();
			}
			else{
				gpu3d_rs::addsolution_warpFrame1_z<<<blocksPerGrid,threadsPerBlock>>>(true, phi, dev_frame0, warped1, u, du);
				hipDeviceSynchronize();
				gpu3d_rs::addsolution_warpFrame1_xy<<<blocksPerGrid,threadsPerBlock>>>(true, warped1, dev_frame0, phi, u, du, confidence);
				hipDeviceSynchronize();
			}
		}
		////////////////////////////////////////////////////////////////////////////////////////

		return;
	}
	void OptFlow_GPU3D_Reshape::run_singleiteration(int level, img_type *frame0, img_type *frame1, int shape[3], ProtocolParameters *params, bool frames_set)
	{
		hipSetDevice(deviceID);

		//Set constant memory
		////////////////////////////////////////////////////////////////////////////////////////
		int nx = shape[0]; int ny = shape[1]; int nz = shape[2];
		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = shape[2]*nslice;
		idx_type n_odd = nstack/2;
		idx_type n_even = nstack-n_odd;

		int threadsPerBlock(params->gpu.threadsPerBlock);
		int blocksPerGrid = (nstack + threadsPerBlock - 1) / (threadsPerBlock);
		int blocksPerGrid2 = (n_even + threadsPerBlock -1) / (threadsPerBlock); //iterate over every second voxel

		idx_type asize1 = nstack*sizeof(*dev_frame0);

		mathtype_solver hx = params->scaling.hx;
		mathtype_solver hy = params->scaling.hy;
		mathtype_solver hz = params->scaling.hz;
		mathtype_solver alpha = params->alpha;
		mathtype_solver omega = params->solver.sor_omega;
		bool precalculate_psi = params->solver.precalculate_psi;
		float localglobal_sigma_data = params->special.localglobal_sigma_data;
		bool use_confidencemap = params->confidence.use_confidencemap;
		bool protect_overlap = params->mosaicing.protect_overlap;
		int overlap = params->mosaicing.overlap;

		if (params->pyramid.alpha_scaling)
			alpha = alpha/pow(params->pyramid.scaling_factor, level);

		int smoothness_id = 0;

			 if (params->solver.flowDerivative_type == "Barron") smoothness_id = 0;
		else if (params->solver.flowDerivative_type == "centraldifference") smoothness_id = 1; //Ershov style
		else if (params->solver.flowDerivative_type == "forwarddifference") smoothness_id = 2; //Liu style
		else std::cout << "Warning! Unknown flowDerivative_type!" << std::endl;

		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nx_c), &nx, sizeof(gpu_const::nx_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::ny_c), &ny, sizeof(gpu_const::ny_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nz_c), &nz, sizeof(gpu_const::nz_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nstack_c), &nstack, sizeof(gpu_const::nstack_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::hx_c), &hx, sizeof(gpu_const::hx_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::hy_c), &hy, sizeof(gpu_const::hy_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::hz_c), &hz, sizeof(gpu_const::hz_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::alpha_c), &alpha, sizeof(gpu_const::alpha_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::omega_c), &omega, sizeof(gpu_const::omega_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::zeroDirichletBoundary_c), &params->constraint.zeroDirichletBoundary,  6*sizeof(int), 0);
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::fixedDirichletBoundary_c), &params->constraint.fixedDirichletBoundary,  6*sizeof(int), 0);
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::lowerIntensityCutoff_c), &(params->constraint.intensityRange[0]), sizeof(gpu_const::lowerIntensityCutoff_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::upperIntensityCutoff_c), &(params->constraint.intensityRange[1]), sizeof(gpu_const::upperIntensityCutoff_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::use_confidencemap_c), &use_confidencemap, sizeof(gpu_const::use_confidencemap_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::precalculated_psi_c), &precalculate_psi, sizeof(gpu_const::precalculated_psi_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::filter_sigma_c), &localglobal_sigma_data, sizeof(gpu_const::filter_sigma_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::protect_overlap_c), &protect_overlap, sizeof(gpu_const::protect_overlap_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::overlap_c), &overlap, sizeof(gpu_const::overlap_c));

		//Reshape the image to even-odd-format and copy to device
		//////////////////////////////////////////
		if(!frames_set)
		{
			img_type *reshaped = (img_type*) malloc(nstack*sizeof(*reshaped));
			reshape_on_host(frame0, reshaped, shape);
			hipMemcpy(dev_frame0, reshaped, asize1, hipMemcpyHostToDevice);
			hipDeviceSynchronize();
			reshape_on_host(frame1, reshaped, shape);
			hipMemcpy(warped1, reshaped, asize1, hipMemcpyHostToDevice);
			hipDeviceSynchronize();
			free(reshaped);
		}
		//////////////////////////////////////////

		////////////////////////////////////////////////////////////////////////////////////////
		//initial warp for frame 1
		gpu3d_rs::addsolution_warpFrame1_z<<<blocksPerGrid,threadsPerBlock>>>(false, phi, dev_frame0, warped1, u, du);
		hipDeviceSynchronize();
		gpu3d_rs::addsolution_warpFrame1_xy<<<blocksPerGrid,threadsPerBlock>>>(false, warped1, dev_frame0, phi, u, du, confidence);
		hipDeviceSynchronize();

		for (int i_inner = 0; i_inner < params->solver.innerIterations; i_inner++)
		{
			if (params->special.localglobal_dataterm)
			{
				gpu3d_rs::update_dataterm<<<blocksPerGrid,threadsPerBlock>>>(dev_frame0, warped1, du, phi);
				hipDeviceSynchronize();
				gpu3d_rs::gaussianfilter3D_x<<<blocksPerGrid,threadsPerBlock>>>(phi, psi);
				hipDeviceSynchronize();
				gpu3d_rs::gaussianfilter3D_y<<<blocksPerGrid,threadsPerBlock>>>(psi, phi);
				hipDeviceSynchronize();
				gpu3d_rs::gaussianfilter3D_z<<<blocksPerGrid,threadsPerBlock>>>(phi, psi);
				hipDeviceSynchronize();
			}
			else if(precalculate_psi)
			{
				gpu3d_rs::update_dataterm<<<blocksPerGrid,threadsPerBlock>>>(dev_frame0, warped1, du, psi);
				//hipDeviceSynchronize();
			}

			//Calculate the smoothness term
			//////////////////////////////////////////////////////////////////////////////
			if      (smoothness_id == 0) gpu3d_rs::update_smoothnessterm_Barron<<<blocksPerGrid,threadsPerBlock>>>(u, du,  phi, adaptivity);
			else if (smoothness_id == 1) gpu3d_rs::update_smoothnessterm_centralDiff<<<blocksPerGrid,threadsPerBlock>>>(u, du,  phi, adaptivity);
			else if (smoothness_id == 2) gpu3d_rs::update_smoothnessterm_forwardDiff<<<blocksPerGrid,threadsPerBlock>>>(u, du,  phi, adaptivity);
			hipDeviceSynchronize();

			//////////////////////////////////////////////////////////////////////////////

			//SOR-Updates with psi calculated on the fly
			//////////////////////////////////////////////////////////////////////////////
			//switching between even and odd
			for (int i_sor = 0; i_sor < 2*params->solver.sorIterations; i_sor++)
			{
				gpu3d_rs::calculate_sorUpdate<<<blocksPerGrid2,threadsPerBlock>>>(i_sor, dev_frame0, warped1, phi, psi, u, du, confidence);
				//hipDeviceSynchronize();
			}
			//////////////////////////////////////////////////////////////////////////////
		}

		gpu3d_rs::addsolution<<<blocksPerGrid,threadsPerBlock>>>(u, du);
		hipDeviceSynchronize();
		////////////////////////////////////////////////////////////////////////////////////////

		return;
	}

	void OptFlow_GPU3D_Reshape::set_flowvector(float* in_vector, int shape[3])
	{
		int nx = shape[0];
		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = nslice*shape[2];
		idx_type asize1 = 3*nstack*sizeof(*u);
		optflow_type *u_tmp;

		//Reorder vector into even first odd second
		u_tmp = (optflow_type*) malloc(3*nstack*sizeof(*u_tmp));

		idx_type n_odd = nstack/2;
		idx_type n_even = nstack-n_odd;

		#pragma omp parallel for
		for (idx_type idx = 0; idx < nstack; idx++)
		{
			idx_type pos;

			if(idx < n_even)
			{
				pos = 2*idx;
				int z = pos/nslice;
				int y = (pos-z*nslice)/nx;

					 if ((nx%2) == 0 && (y%2) != 0 && (z%2) == 0) pos++;
				else if ((nx%2) == 0 && (y%2) == 0 && (z%2) != 0) pos++;
				else if ((nx%2) != 0 && (nslice%2) == 0 && (z%2) != 0) pos++;
			}
			else
			{
				pos = (idx-n_even)*2;
				int z = pos/nslice;
				int y = (pos-z*nslice)/nx;

					 if((nx%2) != 0 && (z%2) == 0) pos++;
				else if((nslice%2) != 0 && (z%2) != 0) pos++;
				else if((nx%2) == 0 && (y%2) == 0 && (z%2) == 0) pos++;
				else if((nx%2) == 0 && (y%2) != 0 && (z%2) != 0) pos++;
			}

			if(pos < nstack)
			{
				u_tmp[idx] = in_vector[pos];
				u_tmp[idx+nstack] = in_vector[pos+nstack];
				u_tmp[idx+2*nstack] = in_vector[pos+2*nstack];
			}
		}

		hipSetDevice(deviceID);
		hipMemcpy(u, u_tmp, asize1, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		return;
	}
	void OptFlow_GPU3D_Reshape::set_confidencemap(float* confidencemap, int shape[3])
	{
		int nx = shape[0];
		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = nslice*shape[2];
		idx_type asize1 = nstack*sizeof(*confidence);
		optflow_type *c_tmp;

		//Reorder vector into even first odd second
		c_tmp = (optflow_type*) malloc(nstack*sizeof(*c_tmp));

		idx_type n_odd = nstack/2;
		idx_type n_even = nstack-n_odd;

		#pragma omp parallel for
		for (idx_type idx = 0; idx < nstack; idx++)
		{
			idx_type pos;

			if(idx < n_even)
			{
				pos = 2*idx;
				int z = pos/nslice;
				int y = (pos-z*nslice)/nx;

					 if ((nx%2) == 0 && (y%2) != 0 && (z%2) == 0) pos++;
				else if ((nx%2) == 0 && (y%2) == 0 && (z%2) != 0) pos++;
				else if ((nx%2) != 0 && (nslice%2) == 0 && (z%2) != 0) pos++;
			}
			else
			{
				pos = (idx-n_even)*2;
				int z = pos/nslice;
				int y = (pos-z*nslice)/nx;

					 if((nx%2) != 0 && (z%2) == 0) pos++;
				else if((nslice%2) != 0 && (z%2) != 0) pos++;
				else if((nx%2) == 0 && (y%2) == 0 && (z%2) == 0) pos++;
				else if((nx%2) == 0 && (y%2) != 0 && (z%2) != 0) pos++;
			}

			if(pos < nstack)
			{
				c_tmp[idx] = confidencemap[pos];
			}
		}

		hipSetDevice(deviceID);
		hipMemcpy(confidence, c_tmp, asize1, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		return;
	}
	void OptFlow_GPU3D_Reshape::set_adaptivitymap(float* adaptivitymap, int shape[3])
	{
		int nx = shape[0];
		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = nslice*shape[2];
		idx_type asize1 = (2*nstack)*sizeof(*adaptivity);
		optflow_type *c_tmp;

		//Reorder vector into even first odd second
		c_tmp = (optflow_type*) malloc((2*nstack)*sizeof(*c_tmp));

		idx_type n_odd = nstack/2;
		idx_type n_even = nstack-n_odd;

		#pragma omp parallel for
		for (idx_type idx = 0; idx < nstack; idx++)
		{
			idx_type pos;

			if(idx < n_even)
			{
				pos = 2*idx;
				int z = pos/nslice;
				int y = (pos-z*nslice)/nx;

					 if ((nx%2) == 0 && (y%2) != 0 && (z%2) == 0) pos++;
				else if ((nx%2) == 0 && (y%2) == 0 && (z%2) != 0) pos++;
				else if ((nx%2) != 0 && (nslice%2) == 0 && (z%2) != 0) pos++;
			}
			else
			{
				pos = (idx-n_even)*2;
				int z = pos/nslice;
				int y = (pos-z*nslice)/nx;

					 if((nx%2) != 0 && (z%2) == 0) pos++;
				else if((nslice%2) != 0 && (z%2) != 0) pos++;
				else if((nx%2) == 0 && (y%2) == 0 && (z%2) == 0) pos++;
				else if((nx%2) == 0 && (y%2) != 0 && (z%2) != 0) pos++;
			}

			if(pos < nstack)
			{
				c_tmp[idx] = adaptivitymap[pos];
				c_tmp[idx+nstack] = adaptivitymap[pos+nstack];
			}
		}

		hipSetDevice(deviceID);
		hipMemcpy(confidence, c_tmp, asize1, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		return;
	}
	void OptFlow_GPU3D_Reshape::get_resultcopy(float* out_vector, int shape[3])
	{
		hipSetDevice(deviceID);

		int nx = shape[0];
		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = nslice*shape[2];
		idx_type asize1 = (3*nstack)*sizeof(*u);

		optflow_type *u_tmp = (optflow_type*) malloc(3*nstack*sizeof(*u_tmp));
		hipMemcpy(u_tmp,u, asize1, hipMemcpyDeviceToHost);

		idx_type n_odd = nstack/2;
		idx_type n_even = nstack-n_odd;

		#pragma omp parallel for
		for (idx_type idx = 0; idx < nstack; idx++)
		{
			idx_type pos;

			if(idx < n_even)
			{
				pos = 2*idx;
				int z = pos/nslice;
				int y = (pos-z*nslice)/nx;

					 if ((nx%2) == 0 && (y%2) != 0 && (z%2) == 0) pos++;
				else if ((nx%2) == 0 && (y%2) == 0 && (z%2) != 0) pos++;
				else if ((nx%2) != 0 && (nslice%2) == 0 && (z%2) != 0) pos++;
			}
			else
			{
				pos = (idx-n_even)*2;
				int z = pos/nslice;
				int y = (pos-z*nslice)/nx;

					 if((nx%2) != 0 && (z%2) == 0) pos++;
				else if((nslice%2) != 0 && (z%2) != 0) pos++;
				else if((nx%2) == 0 && (y%2) == 0 && (z%2) == 0) pos++;
				else if((nx%2) == 0 && (y%2) != 0 && (z%2) != 0) pos++;
			}

			if(pos < nstack)
			{
				out_vector[pos] = u_tmp[idx];
				out_vector[pos+nstack] = u_tmp[idx+nstack];
				out_vector[pos+2*nstack] = u_tmp[idx+2*nstack];
			}
		}

		hipDeviceSynchronize();
		return;
	}
}
