#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <typeinfo>
#include <limits>
#include "optflow_gpu2d.h"
#include "gpu_constants.cuh"
#include "../Derivatives/smoothnessterm_gpu2d.cuh"

/*********************************************************************************************************************************************************
 * Location: Helmholtz-Zentrum fuer Material und Kuestenforschung, Max-Planck-Strasse 1, 21502 Geesthacht
 * Author: Stefan Bruns
 * Contact: bruns@nano.ku.dk
 *
 * License: TBA
 *********************************************************************************************************************************************************/

namespace optflow
{
	namespace gpu2d
	{
		__global__ void gaussianfilter2D_x(optflow_type *input, optflow_type *output)
		{
			//acquire constants
			/////////////////////////////////////////////
			float sigma = gpu_const::filter_sigma_c;

			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;

			bool outofbounds = false;
			idx_type pos = (blockIdx.x*blockDim.x+threadIdx.x);
			if (pos >= nstack) {outofbounds = true; pos = threadIdx.x;}

			int y = pos/nx;
			int x = pos-y*nx;
			/////////////////////////////////////////////

			//Create Gaussian kernel
			///////////////////////////////////////////////////
			int fsize = (int) (3*sigma);
			float kernelsum = 0.0f;
			float valuesum = 0.0f;
			//////////////////////////////////////////////////

			for(int xi=-fsize; xi<=fsize; xi++)
			{
				int x0 = x+xi;

				//reflective boundaries
				if (x0 < 0) x0 = -x0;
				else if (x0 >= nx) x0 = 2*nx-x0-2;

				float kernel_val = expf(-(xi*xi)/(sigma*sigma*2));
				kernelsum += kernel_val;

				__syncthreads();
				valuesum += kernel_val*input[y*nx+x0];
			}

			if(!outofbounds)
				output[pos] = valuesum/kernelsum;

			return;
		}
		__global__ void gaussianfilter2D_y(optflow_type *input, optflow_type *output)
		{
			//acquire constants
			/////////////////////////////////////////////
			float sigma = gpu_const::filter_sigma_c;

			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;

			bool outofbounds = false;
			idx_type pos = (blockIdx.x*blockDim.x+threadIdx.x);
			if (pos >= nstack) {outofbounds = true; pos = threadIdx.x;}

			int y = pos/nx;
			int x = pos-y*nx;
			/////////////////////////////////////////////

			//Create Gaussian kernel
			///////////////////////////////////////////////////
			int fsize = (int) (3*sigma);
			float kernelsum = 0.0f;
			float valuesum = 0.0f;
			//////////////////////////////////////////////////

			for(int yi=-fsize; yi<=fsize; yi++)
			{
				int y0 = y+yi;

				//reflective boundaries
				if (y0 < 0) y0 = -y0;
				else if (y0 >= ny) y0 = 2*ny-y0-2;

				float kernel_val = expf(-(yi*yi)/(sigma*sigma*2));
				kernelsum += kernel_val;

				__syncthreads();
				valuesum += kernel_val*input[y0*nx+x];
			}

			if(!outofbounds)
				output[pos] = valuesum/kernelsum;

			return;
		}

		__device__ __inline__ float interpolate_cubic(float &y0, float &y1, float &y2, float &y3, float &mu)
		{
			float mu2 = mu*mu;

			float a0 = y3-y2-y0+y1;
			float a1 = y0-y1-a0;
			float a2 = y2-y0;
			float a3 = y1;

			return a0*mu*mu2+a1*mu2+a2*mu+a3;
		}

		__global__ void calculate_sorUpdate(int iter, img_type *frame0, img_type *warped1, optflow_type *phi, optflow_type *psi, optflow_type *u, optflow_type *du,
				optflow_type *confidencemap)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			mathtype_solver epsilon_psi_squared = gpu_const::epsilon_psi_squared_c;
			mathtype_solver hx = gpu_const::hx_c;
			mathtype_solver hy = gpu_const::hy_c;
			mathtype_solver alpha = gpu_const::alpha_c;
			mathtype_solver omega = gpu_const::omega_c;

			bool precalculated_psi = gpu_const::precalculated_psi_c;
			bool decoupled_smoothness = gpu_const::decoupled_smoothness_c;
			int slip_depth = gpu_const::slip_depth_c;

			float minIntensity = gpu_const::lowerIntensityCutoff_c;
			float maxIntensity = gpu_const::upperIntensityCutoff_c;

			mathtype_solver alphax = alpha/(hx*hx);
			mathtype_solver alphay = alpha/(hy*hy);

			int spatiotemporalderivative_id = gpu_const::spatiotemporalderivative_id_c;
			bool use_confidencemap = gpu_const::use_confidencemap_c;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;

			//Adjust for even/odd updates in 2D
			///////////////////////////////////
			bool outofbounds = false;
			idx_type pos = 2*(blockIdx.x*blockDim.x+threadIdx.x);
			int y = pos/nx;
			int x = pos-y*nx;

			if((iter%2) == 0){
				if ((nx%2) == 0 && (y%2) != 0 ) {x++; pos++;}}
			else{
				if((nx%2) != 0) {x++; pos++;}
				else if((y%2) == 0){x++; pos++;}}

			if (x >= nx) {x = 0; y++; pos = y*nx+x;}

			if (pos >= nstack || x >= nx){
				outofbounds = true; pos = threadIdx.x;
				y = pos/nx; x = pos-y*nx;
			}
			///////////////////////////////////

			mathtype_solver confidence = 1.0f;
			mathtype_solver psi0 = 0.0f;
			mathtype_solver normalizer_x1 = 0.25f/hx;
			mathtype_solver normalizer_y1 = 0.25f/hy;
			/////////////////////////////////////////////

			//Define the neighbourhood
			/////////////////////////////////////////////
			int yp = y+1;
			int yn = y-1;
			int xp = x+1;
			int xn = x-1;

			mathtype_solver xp_active = 1.0f;
			mathtype_solver xn_active = 1.0f;
			mathtype_solver yp_active = 1.0f;
			mathtype_solver yn_active = 1.0f;

			bool boundary_voxel = false;
			if (xp == nx) {xp_active = 0.0f; xp = x; boundary_voxel = true;}
			else if (xn < 0) {xn_active = 0.0f; xn = x; boundary_voxel = true;}
			if (yp == ny) {yp_active = 0.0f; yp = y; boundary_voxel = true;}
			else if (yn < 0) {yn_active = 0.0f; yn = y; boundary_voxel = true;}

			idx_type npos0 = y*nx + xp;
			idx_type npos1 = y*nx + xn;
			idx_type npos2 = yp*nx + x;
			idx_type npos3 = yn*nx + x;

			mathtype_solver phi_neighbour[8] = {0.0f, 0.0f, 0.0f, 0.0f,0.0f, 0.0f, 0.0f, 0.0f,};
			mathtype_solver du_neighbour[4]  = {0.0f, 0.0f, 0.0f, 0.0f,};
			mathtype_solver dv_neighbour[4]  = {0.0f, 0.0f, 0.0f, 0.0f,};
			/////////////////////////////////////////////

			//Switch to reflective boundary conditions
			/////////////////////////////////////////////
			yp = y+1;
			yn = y-1;
			xp = x+1;
			xn = x-1;

			if (yp == ny) yp -= 2;
			else if (yn == -1) yn = 1;
			if (xp == nx) xp -= 2;
			else if (xn == -1) xn = 1;
			/////////////////////////////////////////////

			/////////////////////////////////////////////
			__syncthreads();
			mathtype_solver phi0 = phi[pos];
			mathtype_solver u0 = u[pos];
			mathtype_solver v0 = u[pos+nstack];
			mathtype_solver du0 = du[pos];
			mathtype_solver dv0 = du[pos+nstack];
			mathtype_solver frame0_val = frame0[pos];

			if (use_confidencemap) confidence = confidencemap[pos];
			if (precalculated_psi) psi0 = psi[pos];
			/////////////////////////////////////////////

			//Read in neighbours with 0-boundaries
			/////////////////////////////////////////////
			phi_neighbour[0] = xp_active*0.5f*(phi[npos0] + phi0);
			du_neighbour[0]  = u[npos0] + du[npos0] - u0;
			dv_neighbour[0]  = u[npos0 + nstack] + du[npos0 + nstack] - v0;

			phi_neighbour[1] = xn_active*0.5f*(phi[npos1] + phi0);
			du_neighbour[1]  = u[npos1] + du[npos1] - u0;
			dv_neighbour[1]  = u[npos1 + nstack] + du[npos1 + nstack] - v0;

			phi_neighbour[2] = yp_active*0.5f*(phi[npos2] + phi0);
			du_neighbour[2]  = u[npos2] + du[npos2] - u0;
			dv_neighbour[2]  = u[npos2 + nstack] + du[npos2 + nstack] - v0;

			phi_neighbour[3] = yn_active*0.5f*(phi[npos3] + phi0);
			du_neighbour[3]  = u[npos3] + du[npos3] - u0;
			dv_neighbour[3]  = u[npos3 + nstack] + du[npos3 + nstack] - v0;

			if(decoupled_smoothness)
			{
				mathtype_solver phi1 = phi[nstack+pos];
				phi_neighbour[4] = xp_active*0.5f*(phi[nstack+npos0] + phi1);
				phi_neighbour[5] = xn_active*0.5f*(phi[nstack+npos1] + phi1);
				phi_neighbour[6] = yp_active*0.5f*(phi[nstack+npos2] + phi1);
				phi_neighbour[7] = yn_active*0.5f*(phi[nstack+npos3] + phi1);
			}
			/////////////////////////////////////////////

			mathtype_solver Idx, Idy, Idt;

			//Calculate spatiotemporal derivatives on the fly
			/////////////////////////////////////////////
			if (spatiotemporalderivative_id < 0)
			{
				Idx = psi[pos+nstack];
				Idy = psi[pos+2*nstack];
				Idt = psi[pos+3*nstack];
			}
			else if (spatiotemporalderivative_id == 1){
				//Horn-Schunck: average of frame1 and frame2, dx-kernel := [-1,1; -1,1], dt: local average
				//////////////////////////////////////////////////////////////////////////////////////////
				//mathtype_solver val00a = frame0[y*nx + x];
				mathtype_solver val10a = frame0[y*nx + xp];
				mathtype_solver val01a = frame0[yp*nx + x];
				mathtype_solver val11a = frame0[yp*nx + xp];

				mathtype_solver val00b = warped1[y*nx + x];
				mathtype_solver val10b = warped1[y*nx + xp];
				mathtype_solver val01b = warped1[yp*nx + x];
				mathtype_solver val11b = warped1[yp*nx + xp];

				Idx = normalizer_x1*((-frame0_val + val10a - val01a + val11a) + (-val00b + val10b - val01b + val11b));
				Idy = normalizer_y1*((-frame0_val - val10a + val01a + val11a) + (-val00b - val10b + val01b + val11b));
				Idt = 0.25f*((val00b+val10b+val01b+val11b)-(frame0_val+val10a+val01a+val11a));
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			else if (spatiotemporalderivative_id == 2){
				//Ershov: average of frame1 and frame2, central difference, dt: forward difference
				//////////////////////////////////////////////////////////////////////////////////////////

				mathtype_solver val_xn_a = frame0[y*nx + xn];
				//mathtype_solver val0a    = frame0[pos];
				mathtype_solver val_xp_a = frame0[y*nx + xp];
				mathtype_solver val_yn_a = frame0[yn*nx + x];
				mathtype_solver val_yp_a = frame0[yp*nx + x];

				mathtype_solver val_xn_b = warped1[y*nx + xn];
				mathtype_solver val0b    = warped1[pos];
				mathtype_solver val_xp_b = warped1[y*nx + xp];
				mathtype_solver val_yn_b = warped1[yn*nx + x];
				mathtype_solver val_yp_b = warped1[yp*nx + x];

				Idx = normalizer_x1*((val_xp_a-val_xn_a)+(val_xp_b-val_xn_b));
				Idy = normalizer_y1*((val_yp_a-val_yn_a)+(val_yp_b-val_yn_b));
				Idt = val0b-frame0_val;
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			else if (spatiotemporalderivative_id == 3){
				//Fourth Order Finite Difference
				//////////////////////////////////////////////////////////////////////////////////////////

				int yp2 = y+2; int yn2 = y-2; int xp2 = x+2; int xn2 = x-2;
				if (yp2 >= ny) yp2 = 2*ny-yp2-2; if (yn2 < 0) yn2 = -yn2;
				if (xp2 >= nx) xp2 = 2*nx-xp2-2; if (xn2 < 0) xn2 = -xn2;

				__syncthreads();
				mathtype_solver val_xn2_a = frame0[y*nx + xn2];
				mathtype_solver val_xn_a = frame0[y*nx + xn];
				//mathtype_solver val0a    = frame0[pos];
				mathtype_solver val_xp_a = frame0[y*nx + xp];
				mathtype_solver val_xp2_a = frame0[y*nx + xp2];
				mathtype_solver val_yn2_a = frame0[yn2*nx + x];
				mathtype_solver val_yn_a = frame0[yn*nx + x];
				mathtype_solver val_yp_a = frame0[yp*nx + x];
				mathtype_solver val_yp2_a = frame0[yp2*nx + x];

				mathtype_solver val_xn2_b = warped1[y*nx + xn2];
				mathtype_solver val_xn_b = warped1[y*nx + xn];
				mathtype_solver val0b    = warped1[pos];
				mathtype_solver val_xp_b = warped1[y*nx + xp];
				mathtype_solver val_xp2_b = warped1[y*nx + xp2];
				mathtype_solver val_yn2_b = warped1[yn2*nx + x];
				mathtype_solver val_yn_b = warped1[yn*nx + x];
				mathtype_solver val_yp_b = warped1[yp*nx + x];
				mathtype_solver val_yp2_b = warped1[yp2*nx + x];

				Idx = normalizer_x1/6.f*((val_xn2_a-8.f*val_xn_a+8.f*val_xp_a-val_xp2_a)+(val_xn2_b-8.f*val_xn_b+8.f*val_xp_b-val_xp2_b));
				Idy = normalizer_y1/6.f*((val_yn2_a-8.f*val_yn_a+8.f*val_yp_a-val_yp2_a)+(val_yn2_b-8.f*val_yn_b+8.f*val_yp_b-val_yp2_b));
				Idt = (val0b-frame0_val);
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			else if (spatiotemporalderivative_id == 4){
				//Sixth Order Finite Difference
				//////////////////////////////////////////////////////////////////////////////////////////

				int yp2 = y+2; int yn2 = y-2; int xp2 = x+2; int xn2 = x-2;
				if (yp2 >= ny) yp2 = 2*ny-yp2-2; if (yn2 < 0) yn2 = -yn2;
				if (xp2 >= nx) xp2 = 2*nx-xp2-2; if (xn2 < 0) xn2 = -xn2;

				int yp3 = y+3; int yn3 = y-3; int xp3 = x+3; int xn3 = x-3;
				if (yp3 >= ny) yp3 = 2*ny-yp3-3; if (yn3 < 0) yn3 = -yn3;
				if (xp3 >= nx) xp3 = 2*nx-xp3-3; if (xn3 < 0) xn3 = -xn3;

				__syncthreads();
				mathtype_solver val_xn3_a = frame0[y*nx + xn3];
				mathtype_solver val_xn2_a = frame0[y*nx + xn2];
				mathtype_solver val_xn_a = frame0[y*nx + xn];
				//mathtype_solver val0a    = frame0[pos];
				mathtype_solver val_xp_a = frame0[y*nx + xp];
				mathtype_solver val_xp2_a = frame0[y*nx + xp2];
				mathtype_solver val_xp3_a = frame0[y*nx + xp3];
				mathtype_solver val_yn3_a = frame0[yn3*nx + x];
				mathtype_solver val_yn2_a = frame0[yn2*nx + x];
				mathtype_solver val_yn_a = frame0[yn*nx + x];
				mathtype_solver val_yp_a = frame0[yp*nx + x];
				mathtype_solver val_yp2_a = frame0[yp2*nx + x];
				mathtype_solver val_yp3_a = frame0[yp3*nx + x];

				mathtype_solver val_xn3_b = warped1[y*nx + xn3];
				mathtype_solver val_xn2_b = warped1[y*nx + xn2];
				mathtype_solver val_xn_b = warped1[y*nx + xn];
				mathtype_solver val0b    = warped1[pos];
				mathtype_solver val_xp_b = warped1[y*nx + xp];
				mathtype_solver val_xp2_b = warped1[y*nx + xp2];
				mathtype_solver val_xp3_b = warped1[y*nx + xp3];
				mathtype_solver val_yn3_b = warped1[yn3*nx + x];
				mathtype_solver val_yn2_b = warped1[yn2*nx + x];
				mathtype_solver val_yn_b = warped1[yn*nx + x];
				mathtype_solver val_yp_b = warped1[yp*nx + x];
				mathtype_solver val_yp2_b = warped1[yp2*nx + x];
				mathtype_solver val_yp3_b = warped1[yp3*nx + x];

				Idx = normalizer_x1/30.f*((-val_xn3_a+9*val_xn2_a-45.f*val_xn_a+45.f*val_xp_a-9*val_xp2_a+val_xp3_a)+(-val_xn3_b+9*val_xn2_b-45.f*val_xn_b+45.f*val_xp_b-9.f*val_xp2_b+val_xp3_b));
				Idy = normalizer_y1/30.f*((-val_yn3_a+9*val_yn2_a-45.f*val_yn_a+45.f*val_yp_a-9*val_yp2_a+val_yp3_a)+(-val_yn3_b+9*val_yn2_b-45.f*val_yn_b+45.f*val_yp_b-9.f*val_yp2_b+val_yp3_b));
				Idt = val0b-frame0_val;
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			else if (spatiotemporalderivative_id == 5){
				//LBM
				//////////////////////////////////////////////////////////////////////////////////////////
				mathtype_solver w2 = 0.02777777777777777f;
				mathtype_solver w1 = 4*w2;

				mathtype_solver val_xnyn_a = frame0[yn*nx + xn];
				mathtype_solver val_xny0_a = frame0[y*nx + xn];
				mathtype_solver val_xnyp_a = frame0[yp*nx + xn];
				mathtype_solver val_x0yn_a = frame0[yn*nx + x];
				mathtype_solver val_x0yp_a = frame0[yp*nx + x];
				mathtype_solver val_xpyn_a = frame0[yn*nx + xp];
				mathtype_solver val_xpy0_a = frame0[y*nx + xp];
				mathtype_solver val_xpyp_a = frame0[yp*nx + xp];

				mathtype_solver val_xnyn_b = warped1[yn*nx + xn];
				mathtype_solver val_xny0_b = warped1[y*nx + xn];
				mathtype_solver val_xnyp_b = warped1[yp*nx + xn];
				mathtype_solver val_x0yn_b = warped1[yn*nx + x];
				mathtype_solver val_x0y0_b = warped1[pos];
				mathtype_solver val_x0yp_b = warped1[yp*nx + x];
				mathtype_solver val_xpyn_b = warped1[yn*nx + xp];
				mathtype_solver val_xpy0_b = warped1[y*nx + xp];
				mathtype_solver val_xpyp_b = warped1[yp*nx + xp];

				Idx = 1.5f*(w1*(val_xpy0_a - val_xny0_a) + w2*( val_xpyn_a +val_xpyp_a -val_xnyn_a -val_xnyp_a));
				Idy = 1.5f*(w1*(val_x0yp_a - val_x0yn_a) + w2*(-val_xpyn_a +val_xpyp_a -val_xnyn_a +val_xnyp_a));

				Idx += 1.5f*(w1*(val_xpy0_b - val_xny0_b) + w2*( val_xpyn_b +val_xpyp_b -val_xnyn_b -val_xnyp_b));
				Idy += 1.5f*(w1*(val_x0yp_b - val_x0yn_b) + w2*(-val_xpyn_b +val_xpyp_b -val_xnyn_b +val_xnyp_b));
				Idt = val_x0y0_b-frame0_val;
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			/////////////////////////////////////////////

			//Intensity constancy:
			/////////////////////////////////////////////
			mathtype_solver J11 = Idx*Idx;
			mathtype_solver J22 = Idy*Idy;
			mathtype_solver J12 = Idx*Idy;
			mathtype_solver J13 = Idx*Idt;
			mathtype_solver J23 = Idy*Idt;
			/////////////////////////////////////////////

			//Calculating data term on the fly doesn't hurt much and saves memory
			//(doesn't work for local global approach)
			////////////////////////////////////////////////////////////////
			if(!precalculated_psi)
			{
				//assuming inner_iterations = 1
				psi0 = Idt;//+Idx*du0+Idy*dv0;
				psi0 *= psi0;
			}
			psi0 = 0.5f/sqrtf(psi0+epsilon_psi_squared);

			if(use_confidencemap) psi0 *= max(0.0f, min(1.0f, confidence));

			//deactivate data term for background:
			if (frame0_val < minIntensity || frame0_val > maxIntensity) psi0 = 0.0f;
			if (slip_depth > 0 && (x < slip_depth || x >= nx-slip_depth || y < slip_depth || y >= ny-slip_depth)) psi0 = 0.0f; //avoid objects getting pinned to the boundary
			////////////////////////////////////////////////////////////////

			//Calculate SOR update
			/////////////////////////////////////////////
			mathtype_solver sumH = alphax*(phi_neighbour[0]+phi_neighbour[1]) + alphay*(phi_neighbour[2]+phi_neighbour[3]);
			mathtype_solver sumU = alphax*(phi_neighbour[0]*du_neighbour[0] + phi_neighbour[1]*du_neighbour[1]) + alphay*(phi_neighbour[2]*du_neighbour[2] + phi_neighbour[3]*du_neighbour[3]);

			mathtype_solver sumH2 = sumH;
			mathtype_solver sumV;
			if(!decoupled_smoothness)
				sumV = alphax*(phi_neighbour[0]*dv_neighbour[0] + phi_neighbour[1]*dv_neighbour[1]) + alphay*(phi_neighbour[2]*dv_neighbour[2] + phi_neighbour[3]*dv_neighbour[3]);
			else
			{
				sumV = alphax*(phi_neighbour[4]*dv_neighbour[0] + phi_neighbour[5]*dv_neighbour[1]) + alphay*(phi_neighbour[6]*dv_neighbour[2] + phi_neighbour[7]*dv_neighbour[3]);
				sumH2 = alphax*(phi_neighbour[4]+phi_neighbour[5]) + alphay*(phi_neighbour[6]+phi_neighbour[7]);
			}
			mathtype_solver next_du, next_dv;

			//SOR-step unless Dirichlet boundary conditions
			////////////////////////////////////////////////////////
			if (boundary_voxel)
			{
				if (    (x == 0 && gpu_const::fixedDirichletBoundary_c[2] == 1) || (x == nx-1 && gpu_const::fixedDirichletBoundary_c[3] == 1)
				     || (y == 0 && gpu_const::fixedDirichletBoundary_c[0] == 1) || (y == ny-1 && gpu_const::fixedDirichletBoundary_c[1] == 1))
				{
					next_du = 0.0f;
					next_dv = 0.0f;
				}
				else
				{
					if ((x == 0 && gpu_const::zeroDirichletBoundary_c[2] == 1) || (x == nx-1 && gpu_const::zeroDirichletBoundary_c[3] == 1))
						next_du = 0.0f; //boundary condition set
					else
						next_du = (1.f-omega)*du0 + omega*(psi0 *(-J13 -J12 * dv0) + sumU)/(psi0*J11 + sumH);

					if ((y == 0 && gpu_const::zeroDirichletBoundary_c[0] == 1) || (y == ny-1 && gpu_const::zeroDirichletBoundary_c[1] == 1))
						next_dv = 0.0f;
					else
						next_dv = (1.f-omega)*dv0 + omega*(psi0 *(-J23 -J12 * next_du) + sumV)/(psi0*J22 + sumH2);
				}
			}
			else
			{
				next_du = (1.f-omega)*du0 + omega*(psi0 *(-J13 -J12 * dv0) + sumU)/(psi0*J11 + sumH);
				next_dv = (1.f-omega)*dv0 + omega*(psi0 *(-J23 -J12 * next_du) + sumV)/(psi0*J22 + sumH2);
			}

			if (gpu_const::protect_overlap_c)
			{
				//extend the Dirichlet boundary inwards for mosaic processing
				int half_overlap = gpu_const::overlap_c/2;

				if (    (x < half_overlap && gpu_const::fixedDirichletBoundary_c[2] == 1) || (x >= nx-1-half_overlap && gpu_const::fixedDirichletBoundary_c[3] == 1)
					 || (y < half_overlap && gpu_const::fixedDirichletBoundary_c[0] == 1) || (y >= ny-1-half_overlap && gpu_const::fixedDirichletBoundary_c[1] == 1))
				{
					next_du = 0.0f;
					next_dv = 0.0f;
				}
			}

			////////////////////////////////////////////////////////

			/////////////////////////////////////////////
			__syncthreads();
			if(!outofbounds)
			{
				du[pos] = next_du;
				du[pos+nstack] = next_dv;
			}
			/////////////////////////////////////////////

			return;
		}

		__global__ void addsolution_warpFrame1_xy(bool rewarp, img_type *warped1, img_type *frame0, img_type *frame1, optflow_type *u, optflow_type *du, optflow_type *confidence){

			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			int outOfBounds_id = gpu_const::outOfBounds_id_c;
			int interpolation_id = gpu_const::warpInterpolation_id_c;
			bool use_confidencemap = gpu_const::use_confidencemap_c;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;

			bool outofbounds = false;
			idx_type pos = (blockIdx.x*blockDim.x+threadIdx.x);
			if (pos >= nstack) {outofbounds = true; pos = threadIdx.x;}

			int z = pos/nslice;
			int y = (pos-z*nslice)/nx;
			int x = pos-z*nslice-y*nx;
			/////////////////////////////////////////////

			/////////////////////////////////////////////
			__syncthreads();
			mathtype_solver u0  = u[pos];
			mathtype_solver v0  = u[pos+nstack];
			mathtype_solver w0 = 0.0f; //should already be warped
			mathtype_solver du0 = du[pos];
			mathtype_solver dv0 = du[pos+nstack];

			if(nz > 1)
				w0 = u[pos+2*nstack]; //for out of bounds checking

			u0 += du0;
			v0 += dv0;

			float x0 = x + u0;
			float y0 = y + v0;
			float z0 = z + w0;

			if(rewarp)
			{
				x0 = x + du0;
				y0 = y + dv0;
				z0 = z;
			}

			//out of bounds?
			////////////////////////
			img_type replace_val = 0.0f;
			bool moved_out = false;

			if (y0 < 0 || x0 < 0 || z0 < 0 || x0 > (nx-1) || y0 > (ny-1) || z0 > (nz-1))
			{
				moved_out = true;
				if (outOfBounds_id == 0) replace_val = frame0[pos];
				else replace_val = gpu_const::nanf_c;

				if (use_confidencemap) confidence[pos] = 0.0f;

				x0 = x; y0 = y; //z0 = z;
			}
			////////////////////////

			int xf = floor(x0);
			int xc = ceil(x0);
			int yf = floor(y0);
			int yc = ceil(y0);

			//extrapolate with zero-gradient
			int xf2 = max(0, xf-1);
			int xc2 = min(xc+1, nx-1);
			int yf2 = max(0, yf-1);
			int yc2 = min(yc+1, ny-1);

			float wx = x0-xf;
			float wy = y0-yf;

			img_type value = 0.0f;

			__syncthreads();
			/////////////////////////////////////////////
			img_type P11 = frame1[z*nslice+ yf*nx + xf];
			img_type P21 = frame1[z*nslice+ yf*nx + xc];
			img_type P12 = frame1[z*nslice+ yc*nx + xf];
			img_type P22 = frame1[z*nslice+ yc*nx + xc];

			if (interpolation_id == 1)
			{
				img_type P10 = frame1[z*nslice + yf2*nx + xf];
				img_type P20 = frame1[z*nslice + yf2*nx + xc];
				img_type P01 = frame1[z*nslice + yf*nx  + xf2];
				img_type P31 = frame1[z*nslice + yf*nx  + xc2];
				img_type P02 = frame1[z*nslice + yc*nx  + xf2];
				img_type P32 = frame1[z*nslice + yc*nx  + xc2];
				img_type P13 = frame1[z*nslice + yc2*nx + xf];
				img_type P23 = frame1[z*nslice + yc2*nx + xc];

				float gtu = gpu2d::interpolate_cubic(P01,P11,P21,P31,wx);
				float gbu = gpu2d::interpolate_cubic(P02,P12,P22,P32,wx);

				float glv = gpu2d::interpolate_cubic(P10,P11,P12,P13,wy);
				float grv = gpu2d::interpolate_cubic(P20,P21,P22,P23,wy);

				float sigma_lr = (1.f-wx)*glv + wx*grv;
				float sigma_bt = (1.f-wy)*gtu + wy*gbu;
				float corr_lrbt = P11*(1.f-wy)*(1.f-wx) + P12*wy*(1.f-wx) + P21*(1.f-wy)*wx + P22*wx*wy;

				value = sigma_lr+sigma_bt-corr_lrbt;
			}
			else
			{
				float glv = (P12-P11)*wy+P11; //left
				float grv = (P22-P21)*wy+P21; //right
				float gtu = (P21-P11)*wx+P11; //top
				float gbu = (P22-P12)*wx+P12; //bottom

				float sigma_lr = (1.f-wx)*glv + wx*grv;
				float sigma_bt = (1.f-wy)*gtu + wy*gbu;
				float corr_lrbt = P11*(1.f-wy)*(1.f-wx) + P12*wy*(1.f-wx) + P21*(1.f-wy)*wx + P22*wx*wy;

				value = sigma_lr+sigma_bt-corr_lrbt;
			}

			if(moved_out) value = replace_val;

			__syncthreads();
			////////////////////////////
			if(!outofbounds)
			{
			warped1[pos] = value;
			u[pos] = u0;
			u[pos+nstack] = v0;
			du[pos] = 0.0f;
			du[pos+nstack] = 0.0f;
			}
			////////////////////////////

			return;
		}
		__global__ void addsolution(optflow_type *u, optflow_type *du)
		{
			//acquire constants
			/////////////////////////////////////////////
			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;

			idx_type pos = (blockIdx.x*blockDim.x+threadIdx.x);
			if (pos >= nstack) {pos = threadIdx.x;}
			/////////////////////////////////////////////

			/////////////////////////////////////////////
			__syncthreads();
			mathtype_solver u0  = u[pos];
			mathtype_solver v0  = u[pos+nstack];
			mathtype_solver du0 = du[pos];
			mathtype_solver dv0 = du[pos+nstack];

			u0 += du0;
			v0 += dv0;

			u[pos] = u0;
			u[pos+nstack] = v0;
			du[pos] = 0.0f;
			du[pos+nstack] = 0.0f;
			////////////////////////////

			return;
		}

		__global__ void update_dataterm(img_type *frame0, img_type *warped1, optflow_type *du, optflow_type *psi)
		{
			//acquire constants and position
			/////////////////////////////////////////////
			int nx = gpu_const::nx_c;
			int ny = gpu_const::ny_c;
			int nz = gpu_const::nz_c;

			int spatiotemporalderivative_id = gpu_const::spatiotemporalderivative_id_c;
			mathtype_solver hx = gpu_const::hx_c;
			mathtype_solver hy = gpu_const::hy_c;

			mathtype_solver normalizer_x1 = 0.25f/hx;
			mathtype_solver normalizer_y1 = 0.25f/hy;

			idx_type nslice = nx*ny;
			idx_type nstack = nz*nslice;

			idx_type pos = (blockIdx.x*blockDim.x+threadIdx.x);
			if (pos >= nstack) pos = threadIdx.x;

			int y = pos/nx;
			int x = pos-y*nx;

			int yp = y+1; int yn = y-1;
			int xp = x+1; int xn = x-1;

			//Reflective boundary conditions
			if (yp == ny) yp -= 2; if (yn < 0) yn = 1;
			if (xp == nx) xp -= 2; if (xn < 0) xn = 1;

			__syncthreads();
			/////////////////////////////////////////////

			float Idx, Idy, Idt;

			mathtype_solver du0 = du[pos];
			mathtype_solver dv0 = du[pos+nstack];
			mathtype_solver frame0_val = frame0[pos];

			//Precalculate spatiotemporal derivatives for local-global
			/////////////////////////////////////////////
			if (abs(spatiotemporalderivative_id) == 1){

				//Horn-Schunck: average of frame1 and frame2, dx-kernel := [-1,1; -1,1], dt: local average
				//////////////////////////////////////////////////////////////////////////////////////////
				mathtype_solver val10a = frame0[y*nx + xp];
				mathtype_solver val01a = frame0[yp*nx + x];
				mathtype_solver val11a = frame0[yp*nx + xp];

				mathtype_solver val00b = warped1[y*nx + x];
				mathtype_solver val10b = warped1[y*nx + xp];
				mathtype_solver val01b = warped1[yp*nx + x];
				mathtype_solver val11b = warped1[yp*nx + xp];

				Idx = normalizer_x1*((-frame0_val + val10a - val01a + val11a) + (-val00b + val10b - val01b + val11b));
				Idy = normalizer_y1*((-frame0_val - val10a + val01a + val11a) + (-val00b - val10b + val01b + val11b));
				Idt = 0.25f*((val00b+val10b+val01b+val11b)-(frame0_val+val10a+val01a+val11a));
				//////////////////////////////////////////////////////////////////////////////////////////

			}
			else if (abs(spatiotemporalderivative_id) == 2){

				//Ershov: average of frame1 and frame2, central difference, dt: forward difference
				//////////////////////////////////////////////////////////////////////////////////////////

				mathtype_solver val_xn_a = frame0[y*nx + xn];
				mathtype_solver val_xp_a = frame0[y*nx + xp];
				mathtype_solver val_yn_a = frame0[yn*nx + x];
				mathtype_solver val_yp_a = frame0[yp*nx + x];

				mathtype_solver val_xn_b = warped1[y*nx + xn];
				mathtype_solver val0b    = warped1[pos];
				mathtype_solver val_xp_b = warped1[y*nx + xp];
				mathtype_solver val_yn_b = warped1[yn*nx + x];
				mathtype_solver val_yp_b = warped1[yp*nx + x];

				Idx = normalizer_x1*((val_xp_a-val_xn_a)+(val_xp_b-val_xn_b));
				Idy = normalizer_y1*((val_yp_a-val_yn_a)+(val_yp_b-val_yn_b));
				Idt = val0b-frame0_val;
				//////////////////////////////////////////////////////////////////////////////////////////

			}
			else if (abs(spatiotemporalderivative_id) == 3){
				//Fourth Order Finite Difference
				//////////////////////////////////////////////////////////////////////////////////////////

				int yp2 = y+2; int yn2 = y-2; int xp2 = x+2; int xn2 = x-2;
				if (yp2 >= ny) yp2 = 2*ny-yp2-2; if (yn2 < 0) yn2 = -yn2;
				if (xp2 >= nx) xp2 = 2*nx-xp2-2; if (xn2 < 0) xn2 = -xn2;

				mathtype_solver val_xn2_a = frame0[y*nx + xn2];
				mathtype_solver val_xn_a = frame0[y*nx + xn];
				//mathtype_solver val0a    = frame0[pos];
				mathtype_solver val_xp_a = frame0[y*nx + xp];
				mathtype_solver val_xp2_a = frame0[y*nx + xp2];
				mathtype_solver val_yn2_a = frame0[yn2*nx + x];
				mathtype_solver val_yn_a = frame0[yn*nx + x];
				mathtype_solver val_yp_a = frame0[yp*nx + x];
				mathtype_solver val_yp2_a = frame0[yp2*nx + x];

				mathtype_solver val_xn2_b = warped1[y*nx + xn2];
				mathtype_solver val_xn_b = warped1[y*nx + xn];
				mathtype_solver val0b    = warped1[pos];
				mathtype_solver val_xp_b = warped1[y*nx + xp];
				mathtype_solver val_xp2_b = warped1[y*nx + xp2];
				mathtype_solver val_yn2_b = warped1[yn2*nx + x];
				mathtype_solver val_yn_b = warped1[yn*nx + x];
				mathtype_solver val_yp_b = warped1[yp*nx + x];
				mathtype_solver val_yp2_b = warped1[yp2*nx + x];

				Idx = normalizer_x1/6.f*((val_xn2_a-8.f*val_xn_a+8.f*val_xp_a-val_xp2_a)+(val_xn2_b-8.f*val_xn_b+8.f*val_xp_b-val_xp2_b));
				Idy = normalizer_y1/6.f*((val_yn2_a-8.f*val_yn_a+8.f*val_yp_a-val_yp2_a)+(val_yn2_b-8.f*val_yn_b+8.f*val_yp_b-val_yp2_b));
				Idt = val0b-frame0_val;
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			else if (abs(spatiotemporalderivative_id) == 4){
				//Sixth Order Finite Difference
				//////////////////////////////////////////////////////////////////////////////////////////

				int yp2 = y+2; int yn2 = y-2; int xp2 = x+2; int xn2 = x-2;
				if (yp2 >= ny) yp2 = 2*ny-yp2-2; if (yn2 < 0) yn2 = -yn2;
				if (xp2 >= nx) xp2 = 2*nx-xp2-2; if (xn2 < 0) xn2 = -xn2;

				int yp3 = y+3; int yn3 = y-3; int xp3 = x+3; int xn3 = x-3;
				if (yp3 >= ny) yp3 = 2*ny-yp3-3; if (yn3 < 0) yn3 = -yn3;
				if (xp3 >= nx) xp3 = 2*nx-xp3-3; if (xn3 < 0) xn3 = -xn3;

				__syncthreads();
				mathtype_solver val_xn3_a = frame0[y*nx + xn3];
				mathtype_solver val_xn2_a = frame0[y*nx + xn2];
				mathtype_solver val_xn_a = frame0[y*nx + xn];
				//mathtype_solver val0a    = frame0[pos];
				mathtype_solver val_xp_a = frame0[y*nx + xp];
				mathtype_solver val_xp2_a = frame0[y*nx + xp2];
				mathtype_solver val_xp3_a = frame0[y*nx + xp3];
				mathtype_solver val_yn3_a = frame0[yn3*nx + x];
				mathtype_solver val_yn2_a = frame0[yn2*nx + x];
				mathtype_solver val_yn_a = frame0[yn*nx + x];
				mathtype_solver val_yp_a = frame0[yp*nx + x];
				mathtype_solver val_yp2_a = frame0[yp2*nx + x];
				mathtype_solver val_yp3_a = frame0[yp3*nx + x];

				mathtype_solver val_xn3_b = warped1[y*nx + xn3];
				mathtype_solver val_xn2_b = warped1[y*nx + xn2];
				mathtype_solver val_xn_b = warped1[y*nx + xn];
				mathtype_solver val0b    = warped1[pos];
				mathtype_solver val_xp_b = warped1[y*nx + xp];
				mathtype_solver val_xp2_b = warped1[y*nx + xp2];
				mathtype_solver val_xp3_b = warped1[y*nx + xp3];
				mathtype_solver val_yn3_b = warped1[yn3*nx + x];
				mathtype_solver val_yn2_b = warped1[yn2*nx + x];
				mathtype_solver val_yn_b = warped1[yn*nx + x];
				mathtype_solver val_yp_b = warped1[yp*nx + x];
				mathtype_solver val_yp2_b = warped1[yp2*nx + x];
				mathtype_solver val_yp3_b = warped1[yp3*nx + x];

				Idx = normalizer_x1/30.f*((-val_xn3_a+9*val_xn2_a-45.f*val_xn_a+45.f*val_xp_a-9*val_xp2_a+val_xp3_a)+(-val_xn3_b+9*val_xn2_b-45.f*val_xn_b+45.f*val_xp_b-9.f*val_xp2_b+val_xp3_b));
				Idy = normalizer_y1/30.f*((-val_yn3_a+9*val_yn2_a-45.f*val_yn_a+45.f*val_yp_a-9*val_yp2_a+val_yp3_a)+(-val_yn3_b+9*val_yn2_b-45.f*val_yn_b+45.f*val_yp_b-9.f*val_yp2_b+val_yp3_b));
				Idt = val0b-frame0_val;
				//////////////////////////////////////////////////////////////////////////////////////////
			}
			else if (abs(spatiotemporalderivative_id) == 5){
				//LBM
				//////////////////////////////////////////////////////////////////////////////////////////
				mathtype_solver w2 = 0.02777777777777777f;
				mathtype_solver w1 = 4*w2;

				mathtype_solver val_xnyn_a = frame0[yn*nx + xn];
				mathtype_solver val_xny0_a = frame0[y*nx + xn];
				mathtype_solver val_xnyp_a = frame0[yp*nx + xn];
				mathtype_solver val_x0yn_a = frame0[yn*nx + x];
				mathtype_solver val_x0yp_a = frame0[yp*nx + x];
				mathtype_solver val_xpyn_a = frame0[yn*nx + xp];
				mathtype_solver val_xpy0_a = frame0[y*nx + xp];
				mathtype_solver val_xpyp_a = frame0[yp*nx + xp];

				mathtype_solver val_xnyn_b = warped1[yn*nx + xn];
				mathtype_solver val_xny0_b = warped1[y*nx + xn];
				mathtype_solver val_xnyp_b = warped1[yp*nx + xn];
				mathtype_solver val_x0yn_b = warped1[yn*nx + x];
				mathtype_solver val_x0y0_b = warped1[pos];
				mathtype_solver val_x0yp_b = warped1[yp*nx + x];
				mathtype_solver val_xpyn_b = warped1[yn*nx + xp];
				mathtype_solver val_xpy0_b = warped1[y*nx + xp];
				mathtype_solver val_xpyp_b = warped1[yp*nx + xp];

				Idx = 1.5f*(w1*(val_xpy0_a - val_xny0_a) + w2*( val_xpyn_a +val_xpyp_a -val_xnyn_a -val_xnyp_a));
				Idy = 1.5f*(w1*(val_x0yp_a - val_x0yn_a) + w2*(-val_xpyn_a +val_xpyp_a -val_xnyn_a +val_xnyp_a));

				Idx += 1.5f*(w1*(val_xpy0_b - val_xny0_b) + w2*( val_xpyn_b +val_xpyp_b -val_xnyn_b -val_xnyp_b));
				Idy += 1.5f*(w1*(val_x0yp_b - val_x0yn_b) + w2*(-val_xpyn_b +val_xpyp_b -val_xnyn_b +val_xnyp_b));
				Idt = val_x0y0_b-frame0_val;
				//////////////////////////////////////////////////////////////////////////////////////////
			}

			mathtype_solver psi0 = Idt+Idx*du0+Idy*dv0;
			psi0 *= psi0;

			__syncthreads();
			psi[pos] = psi0;

			if (spatiotemporalderivative_id < 0)
			{
				psi[pos+nstack] = Idx;
				psi[pos+2*nstack] = Idy;
				psi[pos+3*nstack] = Idt;
			}
			return;
		}

		__global__ void zeroinitialize(optflow_type *u, optflow_type *du, optflow_type *confidence)
		{
			int ndim = gpu_const::ndim_c;
			idx_type nstack = gpu_const::nstack_c;
			bool use_confidencemap = gpu_const::use_confidencemap_c;

			idx_type pos = (blockIdx.x*blockDim.x+threadIdx.x);
			if (pos >= nstack) pos = threadIdx.x;
			__syncthreads();

			if(ndim > 2)
			{
				u[pos] = 0.0f;
				u[pos+nstack] = 0.0f;
				u[pos+2*nstack] = 0.0f;
				du[pos] = 0.0f;
				du[pos+nstack] = 0.0f;
				du[pos+2*nstack] = 0.0f;
			}
			else
			{
				u[pos] = 0.0f;
				u[pos+nstack] = 0.0f;
				du[pos] = 0.0f;
				du[pos+nstack] = 0.0f;
			}
			if (use_confidencemap)
			{
				confidence[pos] = 1.0f;
			}

			return;
		}
		__global__ void reset_du(optflow_type *du)
		{
			int ndim = gpu_const::ndim_c;
			idx_type nstack = gpu_const::nstack_c;

			idx_type pos = (blockIdx.x*blockDim.x+threadIdx.x);
			if (pos >= nstack) pos = threadIdx.x;
			__syncthreads();

			if(ndim > 2)
			{
				du[pos] = 0.0f;
				du[pos+nstack] = 0.0f;
				du[pos+2*nstack] = 0.0f;
			}
			else
			{
				du[pos] = 0.0f;
				du[pos+nstack] = 0.0f;
			}
			return;
		}
	}

	int OptFlow_GPU2D::configure_device(int maxshape[3], ProtocolParameters *params){

		deviceID = params->gpu.deviceID;
		hipSetDevice(deviceID);

		idx_type ndim = 2;
		bool use_confidencemap = params->confidence.use_confidencemap;
		idx_type nstack = maxshape[0]*maxshape[1];
		nstack *= maxshape[2];

		mathtype_solver epsilon_phi_squared = params->smoothness.epsilon_phi;
		epsilon_phi_squared *= epsilon_phi_squared;
		mathtype_solver epsilon_psi_squared = params->solver.epsilon_psi;
		epsilon_psi_squared *= epsilon_psi_squared;
		float nanf = std::numeric_limits<float>::quiet_NaN();

		int outOfBounds_id = 0;
		int warp_interpolation_id = 0;
		int spatiotemporalderivative_id = 0;

			 if (params->warp.outOfBounds_mode == "replace") outOfBounds_id = 0;
		else if (params->warp.outOfBounds_mode == "NaN") outOfBounds_id = 1;
		else std::cout << "Warning! Unknown outOfBounds_mode!" << std::endl;

			 if (params->warp.interpolation_mode == "cubic") warp_interpolation_id = 1;
		else if (params->warp.interpolation_mode == "linear") warp_interpolation_id = 0;
		else std::cout << "Warning! Unknow warp interpolation mode!" << std::endl;

			 if (params->solver.spatiotemporalDerivative_type == "HornSchunck") spatiotemporalderivative_id = 1;
		else if (params->solver.spatiotemporalDerivative_type == "Ershov") spatiotemporalderivative_id = 2; //2nd order
		else if (params->solver.spatiotemporalDerivative_type == "centraldifference") spatiotemporalderivative_id = 2;
		else if (params->solver.spatiotemporalDerivative_type == "Barron") spatiotemporalderivative_id = 3; //4th order
		else if (params->solver.spatiotemporalDerivative_type == "Weickert") spatiotemporalderivative_id = 4; //6th order
		else if (params->solver.spatiotemporalDerivative_type == "LBM") spatiotemporalderivative_id = 5; //6th order
		else {std::cout << "Warning! Unknown spatiotemporal derivative type!" << std::endl;}

		//identify precalculated spatiotemporal derivatives:
		if (params->solver.precalculate_derivatives) spatiotemporalderivative_id *= -1;

		bool anisotropic_smoothness = params->smoothness.anisotropic_smoothness;
		bool decoupled_smoothness = params->smoothness.decoupled_smoothness;
		bool adaptive_smoothness = params->smoothness.adaptive_smoothness;
		bool complementary_smoothness = params->smoothness.complementary_smoothness;

		int slip_depth = params->confidence.slip_depth;

		//check memory requirements
		////////////////////////////////////////////////////
		size_t free_byte, total_byte ;
		hipMemGetInfo( &free_byte, &total_byte ) ;

		int n_optflow = 5;
		int n_img = 2;

		double free_db = (double)free_byte ;
		double expected_usage = 5.*nstack *sizeof(optflow_type);
		expected_usage += 2.*nstack *sizeof(img_type);
		if(params->confidence.use_confidencemap) {expected_usage += nstack *sizeof(optflow_type); n_optflow++;}
		if (params->solver.precalculate_derivatives) {expected_usage += (4*nstack)*sizeof(optflow_type); n_optflow+=4;}
		else if(params->solver.precalculate_psi) {expected_usage += nstack *sizeof(optflow_type); n_optflow++;}
		if(params->warp.rewarp_frame1 == false) {expected_usage += nstack *sizeof(img_type); n_img++;}
		if(params->smoothness.decoupled_smoothness) {expected_usage += nstack *sizeof(optflow_type); n_optflow++;}
		if(params->smoothness.adaptive_smoothness)  {expected_usage += nstack *sizeof(optflow_type); n_optflow++;}

		if (params->mosaicing.mosaic_decomposition && params->mosaicing.max_nstack == -1)
		{
			params->mosaicing.max_nstack = (free_db-(params->mosaicing.memory_buffer*1024*1024))/(n_optflow*sizeof(optflow_type)+n_img*sizeof(img_type));

			if (nstack > params->mosaicing.max_nstack)
			{
				//set nstack to available memory
				expected_usage = expected_usage/nstack*params->mosaicing.max_nstack;
				nstack = params->mosaicing.max_nstack;
				std::cout << "\033[1;31mmax allowed nstack: " << nstack << "\033[0m" << std::endl;
				std::cout << "\033[1;32mGPU memory: " << round(expected_usage/(1024.*1024.)) << " MB out of " << round(free_db/(1024.*1024.)) << " MB\033[0m" << std::endl;
			}
			else
			{
				//deactivate
				params->mosaicing.mosaic_decomposition = false;
				std::cout << "\033[1;32mGPU memory: " << round(expected_usage/(1024.*1024.)) << " MB out of " << round(free_db/(1024.*1024.)) << " MB\033[0m" << std::endl;
			}
		}
		else
		{
			if (expected_usage > free_db){std::cout << "\033[1;31mError! Expected to run out of GPU memory!\033[0m" << std::endl;return 2;}
			else std::cout << "\033[1;32mGPU memory: " << round(expected_usage/(1024.*1024.)) << " MB out of " << round(free_db/(1024.*1024.)) << " MB\033[0m" << std::endl;
		}
		////////////////////////////////////////////////////

		if (params->mosaicing.mosaic_decomposition && params->mosaicing.sequential_approximation == false && params->gpu.n_gpus == 1)
			params->warp.rewarp_frame1 = true; //no reason to keep frame1 in GPU memory (with single GPU)

		//allocate memory and set constant memory
		////////////////////////////////////////////////////
		(optflow_type*) hipMalloc((void**)&u, (ndim*nstack)*sizeof(*u));
		(optflow_type*) hipMalloc((void**)&du, (ndim*nstack)*sizeof(*du));
		if(params->smoothness.decoupled_smoothness) (optflow_type*) hipMalloc((void**)&phi, 2*nstack*sizeof(*phi));
		else (optflow_type*) hipMalloc((void**)&phi, nstack*sizeof(*phi));
		//confidence map or background mask:
		if(params->confidence.use_confidencemap) (optflow_type*) hipMalloc((void**)&confidence, nstack*sizeof(*confidence));
		else (optflow_type*) hipMalloc((void**)&confidence, 0);
		if (params->solver.precalculate_derivatives) (optflow_type*) hipMalloc((void**)&psi, (4*nstack)*sizeof(*psi));
		else if(params->solver.precalculate_psi) (optflow_type*) hipMalloc((void**)&psi, nstack*sizeof(*psi));
		else (optflow_type*) hipMalloc((void**)&psi, 0);
		if(params->smoothness.adaptive_smoothness) (optflow_type*) hipMalloc((void**)&adaptivity, nstack*sizeof(*adaptivity));
		else (optflow_type*) hipMalloc((void**)&adaptivity, 0);

		//using an extra copy to warp from source (rewarp would save a copy)
		(img_type*) hipMalloc((void**)&warped1, nstack*sizeof(*warped1));
		(img_type*) hipMalloc((void**)&dev_frame0, nstack*sizeof(*dev_frame0));
		if (params->warp.rewarp_frame1 == false) (img_type*) hipMalloc((void**)&dev_frame1, nstack*sizeof(*dev_frame1));
		else (img_type*) hipMalloc((void**)&dev_frame1, 0);

		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::ndim_c), &ndim, sizeof(gpu_const::ndim_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nstack_c), &nstack, sizeof(gpu_const::nstack_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::use_confidencemap_c), &use_confidencemap, sizeof(gpu_const::use_confidencemap_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::epsilon_phi_squared_c), &epsilon_phi_squared, sizeof(gpu_const::epsilon_phi_squared_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::epsilon_psi_squared_c), &epsilon_psi_squared, sizeof(gpu_const::epsilon_psi_squared_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::outOfBounds_id_c), &outOfBounds_id, sizeof(gpu_const::outOfBounds_id_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::warpInterpolation_id_c), &warp_interpolation_id, sizeof(gpu_const::warpInterpolation_id_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::spatiotemporalderivative_id_c), &spatiotemporalderivative_id, sizeof(gpu_const::spatiotemporalderivative_id_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nanf_c), &nanf, sizeof(gpu_const::nanf_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::anisotropic_smoothness_c), &anisotropic_smoothness, sizeof(gpu_const::anisotropic_smoothness_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::decoupled_smoothness_c), &decoupled_smoothness, sizeof(gpu_const::decoupled_smoothness_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::adaptive_smoothness_c), &adaptive_smoothness, sizeof(gpu_const::adaptive_smoothness_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::complementary_smoothness_c), &complementary_smoothness, sizeof(gpu_const::complementary_smoothness_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::slip_depth_c), &slip_depth, sizeof(gpu_const::slip_depth_c));
		hipDeviceSynchronize();
		////////////////////////////////////////////////////

		//Initialize arrays
		////////////////////////////////////////////////////
		int threadsPerBlock(params->gpu.threadsPerBlock);
		int blocksPerGrid = (nstack + threadsPerBlock - 1) / (threadsPerBlock);

		gpu2d::zeroinitialize<<<blocksPerGrid,threadsPerBlock>>>(u,du,confidence);
		hipDeviceSynchronize();
		////////////////////////////////////////////////////

		std::string error_string = (std::string) hipGetErrorString(hipGetLastError());
		if (error_string != "no error")
		{
			std::cout << "Device Variable Copying: " << error_string << std::endl;
			return 1;
		}

		return 0;
	}
	void OptFlow_GPU2D::free_device(){
		hipSetDevice(deviceID);

		hipFree(u);
		hipFree(du);
		hipFree(phi);
		hipFree(psi);
		hipFree(confidence);
		hipFree(dev_frame0);
		hipFree(dev_frame1);
		hipFree(warped1);
		hipFree(adaptivity);
	}

	void OptFlow_GPU2D::run_outeriterations(int level, img_type *frame0, img_type *frame1, int shape[3], ProtocolParameters *params, bool resumed_state, bool frames_set)
	{
		hipSetDevice(deviceID);

		//Set constant memory
		////////////////////////////////////////////////////////////////////////////////////////
		int nx = shape[0]; int ny = shape[1]; int nz = shape[2];
		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = shape[2]*nslice;
		idx_type n_odd = nstack/2;
		idx_type n_even = nstack-n_odd;

		int threadsPerBlock(params->gpu.threadsPerBlock);
		int blocksPerGrid = (nstack + threadsPerBlock - 1) / (threadsPerBlock);
		int blocksPerGrid2 = (n_even + threadsPerBlock -1) / (threadsPerBlock); //iterate over every second voxel

		idx_type asize1 = nstack*sizeof(*dev_frame0);

		mathtype_solver hx = params->scaling.hx;
		mathtype_solver hy = params->scaling.hy;
		mathtype_solver alpha = params->alpha;
		mathtype_solver omega = params->solver.sor_omega;
		bool precalculate_psi = params->solver.precalculate_psi;
		float localglobal_sigma_data = params->special.localglobal_sigma_data;
		bool rewarp = params->warp.rewarp_frame1;
		bool use_confidencemap = params->confidence.use_confidencemap;
		bool protect_overlap = params->mosaicing.protect_overlap;
		int overlap = params->mosaicing.overlap;

		if (params->pyramid.alpha_scaling)
			alpha = alpha/pow(params->pyramid.scaling_factor, level);

		int smoothness_id = 0;

			 if (params->solver.flowDerivative_type == "Barron") smoothness_id = 0;
		else if (params->solver.flowDerivative_type == "centraldifference") smoothness_id = 1; //Ershov style
		else if (params->solver.flowDerivative_type == "forwarddifference") smoothness_id = 2; //Liu style
		else if (params->solver.flowDerivative_type == "LBM") smoothness_id = 3; //Lattice Boltzmann style
		else if (params->solver.flowDerivative_type == "Weickert") smoothness_id = 4;
		else if (params->solver.flowDerivative_type == "Leclaire_FIII") smoothness_id = 5;
		else if (params->solver.flowDerivative_type == "Leclaire_FIV") smoothness_id = 6;
		else if (params->solver.flowDerivative_type == "Scharr3") smoothness_id = 7;
		else if (params->solver.flowDerivative_type == "Sobel") smoothness_id = 8;
		else std::cout << "Warning! Unknown flowDerivative_type!" << std::endl;

		if (!resumed_state)
		{
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nx_c), &nx, sizeof(gpu_const::nx_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::ny_c), &ny, sizeof(gpu_const::ny_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nz_c), &nz, sizeof(gpu_const::nz_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nstack_c), &nstack, sizeof(gpu_const::nstack_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::hx_c), &hx, sizeof(gpu_const::hx_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::hy_c), &hy, sizeof(gpu_const::hy_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::alpha_c), &alpha, sizeof(gpu_const::alpha_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::omega_c), &omega, sizeof(gpu_const::omega_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::zeroDirichletBoundary_c), &params->constraint.zeroDirichletBoundary,  6*sizeof(int), 0);
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::fixedDirichletBoundary_c), &params->constraint.fixedDirichletBoundary,  6*sizeof(int), 0);
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::lowerIntensityCutoff_c), &(params->constraint.intensityRange[0]), sizeof(gpu_const::lowerIntensityCutoff_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::upperIntensityCutoff_c), &(params->constraint.intensityRange[1]), sizeof(gpu_const::upperIntensityCutoff_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::use_confidencemap_c), &use_confidencemap, sizeof(gpu_const::use_confidencemap_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::precalculated_psi_c), &precalculate_psi, sizeof(gpu_const::precalculated_psi_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::filter_sigma_c), &localglobal_sigma_data, sizeof(gpu_const::filter_sigma_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::protect_overlap_c), &protect_overlap, sizeof(gpu_const::protect_overlap_c));
			hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::overlap_c), &overlap, sizeof(gpu_const::overlap_c));

			if(!frames_set)
			{
				hipMemcpy(dev_frame0, frame0, asize1, hipMemcpyHostToDevice);
				if(!rewarp) hipMemcpy(dev_frame1, frame1, asize1, hipMemcpyHostToDevice);
				else hipMemcpy(phi, frame1, asize1, hipMemcpyHostToDevice); //keep dev_frame1 in phi for initial warp
				hipDeviceSynchronize();
			}
			////////////////////////////////////////////////////////////////////////////////////////

			//initial warp for frame 1
			if(!rewarp) gpu2d::addsolution_warpFrame1_xy<<<blocksPerGrid,threadsPerBlock>>>(false, warped1, dev_frame0, dev_frame1, u, du, confidence);
			else gpu2d::addsolution_warpFrame1_xy<<<blocksPerGrid,threadsPerBlock>>>(false, warped1, dev_frame0, phi, u, du, confidence);
			hipDeviceSynchronize();
		}

		for (int i_outer = 0; i_outer < params->solver.outerIterations; i_outer++)
		{
			std::cout << "level " << level << " (" << nx << "," << ny << "," << nz << "): " << (i_outer+1) << " \r";
			std::cout.flush();

			for (int i_inner = 0; i_inner < params->solver.innerIterations; i_inner++)
			{
				//gpu2d::reset_du<<<blocksPerGrid,threadsPerBlock>>>(du);
				//hipDeviceSynchronize();

				if(precalculate_psi)
				{
					gpu2d::update_dataterm<<<blocksPerGrid,threadsPerBlock>>>(dev_frame0, warped1, du, psi);
					hipDeviceSynchronize();
				}
				if (params->special.localglobal_dataterm)
				{
					gpu2d::gaussianfilter2D_x<<<blocksPerGrid,threadsPerBlock>>>(psi, phi);
					hipDeviceSynchronize();
					gpu2d::gaussianfilter2D_y<<<blocksPerGrid,threadsPerBlock>>>(phi, psi);
					hipDeviceSynchronize();
				}

				//Calculate the smoothness term
				//////////////////////////////////////////////////////////////////////////////
				if      (smoothness_id == 0) gpu2d::update_smoothnessterm_Barron<<<blocksPerGrid,threadsPerBlock>>>(u, du,  phi, adaptivity);
				else if (smoothness_id == 1) gpu2d::update_smoothnessterm_centralDiff<<<blocksPerGrid,threadsPerBlock>>>(u, du,  phi, adaptivity);
				else if (smoothness_id == 2) gpu2d::update_smoothnessterm_forwardDiff<<<blocksPerGrid,threadsPerBlock>>>(u, du,  phi, adaptivity);
				else if (smoothness_id == 3) gpu2d::update_smoothnessterm_LBM<<<blocksPerGrid,threadsPerBlock>>>(u, du, phi, adaptivity);
				else if (smoothness_id == 4) gpu2d::update_smoothnessterm_Weickert<<<blocksPerGrid,threadsPerBlock>>>(u, du, phi, adaptivity);
				else if (smoothness_id == 5) gpu2d::update_smoothnessterm_Leclaire_FIII<<<blocksPerGrid,threadsPerBlock>>>(u, du, phi, adaptivity);
				else if (smoothness_id == 6) gpu2d::update_smoothnessterm_Leclaire_FIV<<<blocksPerGrid,threadsPerBlock>>>(u, du, phi, adaptivity);
				else if (smoothness_id == 7) gpu2d::update_smoothnessterm_Scharr3<<<blocksPerGrid,threadsPerBlock>>>(u, du, phi, adaptivity);
				else if (smoothness_id == 8) gpu2d::update_smoothnessterm_Sobel<<<blocksPerGrid,threadsPerBlock>>>(u, du, phi, adaptivity);
				hipDeviceSynchronize();
				//////////////////////////////////////////////////////////////////////////////

				//SOR-Updates with psi calculated on the fly
				//////////////////////////////////////////////////////////////////////////////
				//switching between even and odd
				for (int i_sor = 0; i_sor < 2*params->solver.sorIterations; i_sor++)
				{
					//reset on first sor is optional... deprecated for now, previous result is a better guess
					gpu2d::calculate_sorUpdate<<<blocksPerGrid2,threadsPerBlock>>>(i_sor, dev_frame0, warped1, phi, psi, u, du, confidence);
					hipDeviceSynchronize();
				}
				//////////////////////////////////////////////////////////////////////////////
			}

			if (rewarp)
			{
				gpu2d::addsolution_warpFrame1_xy<<<blocksPerGrid,threadsPerBlock>>>(true, phi, dev_frame0, warped1, u, du, confidence);
				hipDeviceSynchronize();
				hipMemcpy(warped1, phi, asize1, hipMemcpyDeviceToDevice);
			}
			else gpu2d::addsolution_warpFrame1_xy<<<blocksPerGrid,threadsPerBlock>>>(false, warped1, dev_frame0, dev_frame1, u, du, confidence);
			hipDeviceSynchronize();
		}

		return;
	}
	void OptFlow_GPU2D::run_singleiteration(int level, img_type *frame0, img_type *frame1, int shape[3], ProtocolParameters *params, bool frames_set)
	{
		hipSetDevice(deviceID);
		//for multiGPU rewarp option needs to be reactivated!!

		//Set constant memory
		////////////////////////////////////////////////////////////////////////////////////////
		int nx = shape[0]; int ny = shape[1]; int nz = shape[2];
		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = shape[2]*nslice;
		idx_type n_odd = nstack/2;
		idx_type n_even = nstack-n_odd;

		int threadsPerBlock(params->gpu.threadsPerBlock);
		int blocksPerGrid = (nstack + threadsPerBlock - 1) / (threadsPerBlock);
		int blocksPerGrid2 = (n_even + threadsPerBlock -1) / (threadsPerBlock); //iterate over every second voxel

		idx_type asize1 = nstack*sizeof(*dev_frame0);

		mathtype_solver hx = params->scaling.hx;
		mathtype_solver hy = params->scaling.hy;
		mathtype_solver alpha = params->alpha;
		mathtype_solver omega = params->solver.sor_omega;
		bool precalculate_psi = params->solver.precalculate_psi;
		float localglobal_sigma_data = params->special.localglobal_sigma_data;
		bool use_confidencemap = params->confidence.use_confidencemap;
		bool protect_overlap = params->mosaicing.protect_overlap;
		int overlap = params->mosaicing.overlap;

		if (params->pyramid.alpha_scaling)
			alpha = alpha/pow(params->pyramid.scaling_factor, level);

		int smoothness_id = 0;

			 if (params->solver.flowDerivative_type == "Barron") smoothness_id = 0;
		else if (params->solver.flowDerivative_type == "centraldifference") smoothness_id = 1; //Ershov style
		else if (params->solver.flowDerivative_type == "forwarddifference") smoothness_id = 2; //Liu style
		else if (params->solver.flowDerivative_type == "LBM") smoothness_id = 3; //Lattice Boltzmann style
		else if (params->solver.flowDerivative_type == "Weickert") smoothness_id = 4;
		else if (params->solver.flowDerivative_type == "Leclaire_FIII") smoothness_id = 5;
		else if (params->solver.flowDerivative_type == "Leclaire_FIV") smoothness_id = 6;
		else if (params->solver.flowDerivative_type == "Scharr3") smoothness_id = 7;
		else if (params->solver.flowDerivative_type == "Sobel") smoothness_id = 8;
		else std::cout << "Warning! Unknown flowDerivative_type!" << std::endl;

		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nx_c), &nx, sizeof(gpu_const::nx_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::ny_c), &ny, sizeof(gpu_const::ny_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nz_c), &nz, sizeof(gpu_const::nz_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::nstack_c), &nstack, sizeof(gpu_const::nstack_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::hx_c), &hx, sizeof(gpu_const::hx_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::hy_c), &hy, sizeof(gpu_const::hy_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::alpha_c), &alpha, sizeof(gpu_const::alpha_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::omega_c), &omega, sizeof(gpu_const::omega_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::zeroDirichletBoundary_c), &params->constraint.zeroDirichletBoundary,  6*sizeof(int), 0);
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::fixedDirichletBoundary_c), &params->constraint.fixedDirichletBoundary,  6*sizeof(int), 0);
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::lowerIntensityCutoff_c), &(params->constraint.intensityRange[0]), sizeof(gpu_const::lowerIntensityCutoff_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::upperIntensityCutoff_c), &(params->constraint.intensityRange[1]), sizeof(gpu_const::upperIntensityCutoff_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::use_confidencemap_c), &use_confidencemap, sizeof(gpu_const::use_confidencemap_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::precalculated_psi_c), &precalculate_psi, sizeof(gpu_const::precalculated_psi_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::filter_sigma_c), &localglobal_sigma_data, sizeof(gpu_const::filter_sigma_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::protect_overlap_c), &protect_overlap, sizeof(gpu_const::protect_overlap_c));
		hipMemcpyToSymbol(HIP_SYMBOL(gpu_const::overlap_c), &overlap, sizeof(gpu_const::overlap_c));

		if (!frames_set)
		{
			hipMemcpy(dev_frame0, frame0, asize1, hipMemcpyHostToDevice);
			hipMemcpy(phi, frame1, asize1, hipMemcpyHostToDevice); //keep dev_frame1 in phi for initial warp
		}
		hipDeviceSynchronize();
		////////////////////////////////////////////////////////////////////////////////////////

		//initial warp for frame 1
		gpu2d::addsolution_warpFrame1_xy<<<blocksPerGrid,threadsPerBlock>>>(false, warped1, dev_frame0, phi, u, du, confidence);
		hipDeviceSynchronize();

		for (int i_inner = 0; i_inner < params->solver.innerIterations; i_inner++)
		{
			if(precalculate_psi)
			{
				gpu2d::update_dataterm<<<blocksPerGrid,threadsPerBlock>>>(dev_frame0, warped1, du, psi);
				hipDeviceSynchronize();
			}
			if (params->special.localglobal_dataterm)
			{
				gpu2d::gaussianfilter2D_x<<<blocksPerGrid,threadsPerBlock>>>(psi, phi);
				hipDeviceSynchronize();
				gpu2d::gaussianfilter2D_y<<<blocksPerGrid,threadsPerBlock>>>(phi, psi);
				hipDeviceSynchronize();
			}

			//Calculate the smoothness term
			//////////////////////////////////////////////////////////////////////////////
			if      (smoothness_id == 0) gpu2d::update_smoothnessterm_Barron<<<blocksPerGrid,threadsPerBlock>>>(u, du,  phi, adaptivity);
			else if (smoothness_id == 1) gpu2d::update_smoothnessterm_centralDiff<<<blocksPerGrid,threadsPerBlock>>>(u, du,  phi, adaptivity);
			else if (smoothness_id == 2) gpu2d::update_smoothnessterm_forwardDiff<<<blocksPerGrid,threadsPerBlock>>>(u, du,  phi, adaptivity);
			else if (smoothness_id == 3) gpu2d::update_smoothnessterm_LBM<<<blocksPerGrid,threadsPerBlock>>>(u, du, phi, adaptivity);
			else if (smoothness_id == 4) gpu2d::update_smoothnessterm_Weickert<<<blocksPerGrid,threadsPerBlock>>>(u, du, phi, adaptivity);
			else if (smoothness_id == 5) gpu2d::update_smoothnessterm_Leclaire_FIII<<<blocksPerGrid,threadsPerBlock>>>(u, du, phi, adaptivity);
			else if (smoothness_id == 6) gpu2d::update_smoothnessterm_Leclaire_FIV<<<blocksPerGrid,threadsPerBlock>>>(u, du, phi, adaptivity);
			else if (smoothness_id == 7) gpu2d::update_smoothnessterm_Scharr3<<<blocksPerGrid,threadsPerBlock>>>(u, du, phi, adaptivity);
			else if (smoothness_id == 8) gpu2d::update_smoothnessterm_Sobel<<<blocksPerGrid,threadsPerBlock>>>(u, du, phi, adaptivity);
			hipDeviceSynchronize();
			//////////////////////////////////////////////////////////////////////////////

			//SOR-Updates with psi calculated on the fly
			//////////////////////////////////////////////////////////////////////////////
			//switching between even and odd
			for (int i_sor = 0; i_sor < 2*params->solver.sorIterations; i_sor++)
			{
				//reset on first sor is optional... deprecated for now, previous result is a better guess
				gpu2d::calculate_sorUpdate<<<blocksPerGrid2,threadsPerBlock>>>(i_sor, dev_frame0, warped1, phi, psi, u, du, confidence);
				hipDeviceSynchronize();
			}
			//////////////////////////////////////////////////////////////////////////////
		}

		gpu2d::addsolution<<<blocksPerGrid,threadsPerBlock>>>(u, du);
		hipDeviceSynchronize();

		return;
	}

	void OptFlow_GPU2D::set_frames(float* frame0, float *frame1, int shape[3], std::vector<int> &boundaries, bool rewarp)
	{
		hipSetDevice(deviceID);

		int nx = shape[0];

		if (boundaries[0] == 0 && boundaries[3] == nx)
		{
			idx_type offset = boundaries[1]*nx;
			idx_type nslice = boundaries[3]*(boundaries[4]-boundaries[1]);
			idx_type asize1 = nslice*sizeof(*dev_frame0);

			hipMemcpyAsync(dev_frame0, frame0 + offset, asize1, hipMemcpyHostToDevice);
			if(!rewarp) hipMemcpyAsync(dev_frame1, frame1 + offset, asize1, hipMemcpyHostToDevice);
			else hipMemcpyAsync(phi, frame1 + offset, asize1, hipMemcpyHostToDevice);
		}
		else
		{
			idx_type nrow_target = (boundaries[3]-boundaries[0]);
			idx_type nrows = boundaries[4]-boundaries[1];
			idx_type asize1 = nrow_target*sizeof(*dev_frame0);

			for (int y = 0; y < nrows; y++)
			{
				idx_type offset_target = y*nrow_target;
				idx_type offset_source = (y+boundaries[1])*nx + boundaries[0];

				hipMemcpyAsync(dev_frame0+offset_target, frame0+offset_source, asize1, hipMemcpyHostToDevice);
				if(!rewarp) hipMemcpyAsync(dev_frame1+offset_target, frame1+offset_source, asize1, hipMemcpyHostToDevice);
				else hipMemcpyAsync(phi+offset_target, frame1+offset_source, asize1, hipMemcpyHostToDevice);
			}
		}
		hipDeviceSynchronize();
		return;
	}
	void OptFlow_GPU2D::set_flowvector(float* in_vector, int shape[3])
	{
		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = nslice*shape[2];
		idx_type asize1 = 2*nstack*sizeof(*u);
		optflow_type *u_tmp;

		if(typeid(float) != typeid(optflow_type))
		{
			u_tmp = (optflow_type*) malloc(2*nstack*sizeof(*u_tmp));

			#pragma omp parallel for
			for (idx_type pos = 0; pos < nstack; pos++)
			{
				u_tmp[pos] = in_vector[pos];
				u_tmp[pos+nstack] = in_vector[pos+nstack];
			}
		}
		else
			u_tmp = in_vector;

		hipSetDevice(deviceID);
		hipMemcpy(u, u_tmp, asize1, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		return;
	}
	void OptFlow_GPU2D::set_flowvector(optflow_type* in_vector, int shape[3], std::vector<int> &boundaries)
	{
		hipSetDevice(deviceID);

		int nx = shape[0];
		long long int nslice_full = nx*shape[1];
		long long int nstack_full = shape[2]*nslice_full;

		if (boundaries[0] == 0 && boundaries[3] == nx)
		{
			idx_type offset = boundaries[1]*nx;
			idx_type nslice = boundaries[3]*(boundaries[4]-boundaries[1]);
			idx_type asize1 = nslice*sizeof(*u);

			hipMemcpyAsync(u, in_vector + offset, asize1, hipMemcpyHostToDevice);
			hipMemcpyAsync(u+nslice, in_vector + offset +nstack_full, asize1, hipMemcpyHostToDevice);
		}
		else
		{
			idx_type nrow_target = (boundaries[3]-boundaries[0]);
			idx_type nrows = boundaries[4]-boundaries[1];
			idx_type nslice_target = nrow_target*nrows;
			idx_type asize1 = nrow_target*sizeof(*u);

			for (int y = 0; y < nrows; y++)
			{
				idx_type offset_target = y*nrow_target;
				idx_type offset_source = (y+boundaries[1])*nx + boundaries[0];

				hipMemcpyAsync(u+offset_target, in_vector+offset_source, asize1, hipMemcpyHostToDevice);
				hipMemcpyAsync(u+offset_target+nslice_target, in_vector+offset_source+nstack_full, asize1, hipMemcpyHostToDevice);
			}
		}
		hipDeviceSynchronize();
		return;
	}
	void OptFlow_GPU2D::set_confidencemap(float* confidencemap, int shape[3])
	{
		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = nslice*shape[2];
		idx_type asize1 = nstack*sizeof(*confidence);
		optflow_type *c_tmp;

		if(typeid(float) != typeid(optflow_type))
		{
			c_tmp = (optflow_type*) malloc(nstack*sizeof(*c_tmp));

			#pragma omp parallel for
			for (idx_type pos = 0; pos < nstack; pos++)
			{
				c_tmp[pos] = confidencemap[pos];
			}
		}
		else
			c_tmp = confidencemap;

		hipSetDevice(deviceID);
		hipMemcpy(confidence, c_tmp, asize1, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		return;
	}
	void OptFlow_GPU2D::set_confidencemap(optflow_type* confidencemap, int shape[3], std::vector<int> &boundaries)
	{
		hipSetDevice(deviceID);

		int nx = shape[0];

		if (boundaries[0] == 0 && boundaries[3] == nx)
		{
			idx_type offset = boundaries[1]*nx;
			idx_type nslice = boundaries[3]*(boundaries[4]-boundaries[1]);
			idx_type asize1 = nslice*sizeof(*confidence);

			hipMemcpy(confidence, confidencemap + offset, asize1, hipMemcpyHostToDevice);
		}
		else
		{
			idx_type nrow_target = (boundaries[3]-boundaries[0]);
			idx_type nrows = boundaries[4]-boundaries[1];
			idx_type asize1 = nrow_target*sizeof(*confidence);

			for (int y = 0; y < nrows; y++)
			{
				idx_type offset_target = y*nrow_target;
				idx_type offset_source = (y+boundaries[1])*nx + boundaries[0];

				hipMemcpyAsync(confidence+offset_target, confidencemap+offset_source, asize1, hipMemcpyHostToDevice);
			}
		}
		hipDeviceSynchronize();
		return;
	}
	void OptFlow_GPU2D::set_adaptivitymap(float* adaptivitymap, int shape[3])
	{
		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = nslice*shape[2];
		idx_type asize1 = (nstack)*sizeof(*adaptivity);
		optflow_type *c_tmp;

		if(typeid(float) != typeid(optflow_type))
		{
			c_tmp = (optflow_type*) malloc(nstack*sizeof(*c_tmp));

			#pragma omp parallel for
			for (idx_type pos = 0; pos < nstack; pos++)
			{
				c_tmp[pos] = adaptivitymap[pos];
			}
		}
		else
			c_tmp = adaptivitymap;

		hipSetDevice(deviceID);
		hipMemcpy(adaptivity, c_tmp, asize1, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		return;
	}
	void OptFlow_GPU2D::set_adaptivitymap(optflow_type* adaptivitymap, int shape[3], std::vector<int> &boundaries)
	{
		hipSetDevice(deviceID);

		int nx = shape[0];

		if (boundaries[0] == 0 && boundaries[3] == nx)
		{
			idx_type offset = boundaries[1]*nx;
			idx_type nslice = boundaries[3]*(boundaries[4]-boundaries[1]);
			idx_type asize1 = nslice*sizeof(*adaptivity);

			hipMemcpy(adaptivity, adaptivitymap + offset, asize1, hipMemcpyHostToDevice);
		}
		else
		{
			idx_type nrow_target = (boundaries[3]-boundaries[0]);
			idx_type nrows = boundaries[4]-boundaries[1];
			idx_type asize1 = nrow_target*sizeof(*adaptivity);

			for (int y = 0; y < nrows; y++)
			{
				idx_type offset_target = y*nrow_target;
				idx_type offset_source = (y+boundaries[1])*nx + boundaries[0];

				hipMemcpyAsync(adaptivity+offset_target, adaptivitymap+offset_source, asize1, hipMemcpyHostToDevice);
			}
		}
		hipDeviceSynchronize();
		return;
	}
	void OptFlow_GPU2D::get_resultcopy(float* out_vector, int shape[3])
	{
		hipSetDevice(deviceID);

		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = nslice*shape[2];
		idx_type asize1 = 2*nstack*sizeof(*u);

		if(typeid(float) != typeid(optflow_type))
		{
			optflow_type *u_tmp = (optflow_type*) malloc(2*nstack*sizeof(*u_tmp));

			hipMemcpy(u_tmp,u, asize1, hipMemcpyDeviceToHost);

			#pragma omp parallel for
			for (idx_type pos = 0; pos < nstack; pos++)
			{
				out_vector[pos] = u_tmp[pos];
				out_vector[pos+nstack] = u_tmp[pos+nstack];
			}
		}
		else
			hipMemcpy(out_vector,u, asize1, hipMemcpyDeviceToHost);

		hipDeviceSynchronize();
		return;
	}
	void OptFlow_GPU2D::get_resultcopy(optflow_type* out_vector, int shape[3], std::vector<int> &boundaries)
	{
		hipSetDevice(deviceID);

		int nx = shape[0];
		long long int nslice_full = nx*shape[1];
		long long int nstack_full = shape[2]*nslice_full;

		if (boundaries[0] == 0 && boundaries[3] == nx)
		{
			idx_type offset = boundaries[1]*nx;
			idx_type nslice = boundaries[3]*(boundaries[4]-boundaries[1]);
			idx_type asize1 = nslice*sizeof(*u);

			hipMemcpyAsync(out_vector + offset, u, asize1, hipMemcpyDeviceToHost);
			hipMemcpyAsync(out_vector + offset + nstack_full, u + nslice, asize1, hipMemcpyDeviceToHost);
		}
		else
		{
			idx_type nrow_source = (boundaries[3]-boundaries[0]);
			idx_type nrows = boundaries[4]-boundaries[1];
			idx_type nslice_source = nrow_source*nrows;
			idx_type asize1 = nrow_source*sizeof(*u);

			for (int y = 0; y < nrows; y++)
			{
				idx_type offset_source = y*nrow_source;
				idx_type offset_target = (y+boundaries[1])*nx + boundaries[0];

				hipMemcpyAsync(out_vector+offset_target,u+offset_source,  asize1, hipMemcpyDeviceToHost);
				hipMemcpyAsync(out_vector+offset_target+nstack_full,u+offset_source+nslice_source, asize1, hipMemcpyDeviceToHost);
			}
		}
		hipDeviceSynchronize();

		return;
	}
	void OptFlow_GPU2D::get_psimap(float* outimg, int shape[3])
	{
		hipSetDevice(deviceID);

		idx_type nslice = shape[0]*shape[1];
		idx_type nstack = nslice*shape[2];
		idx_type asize1 = nstack*sizeof(*psi);

		if(typeid(float) != typeid(optflow_type))
		{
			optflow_type *u_tmp = (optflow_type*) malloc(2*nstack*sizeof(*u_tmp));

			hipMemcpy(u_tmp,psi, asize1, hipMemcpyDeviceToHost);

			#pragma omp parallel for
			for (idx_type pos = 0; pos < nstack; pos++)
			{
				outimg[pos] = u_tmp[pos];
				outimg[pos+nstack] = u_tmp[pos+nstack];
			}
		}
		else
			hipMemcpy(outimg,psi, asize1, hipMemcpyDeviceToHost);

		hipDeviceSynchronize();
		return;
	}
}
